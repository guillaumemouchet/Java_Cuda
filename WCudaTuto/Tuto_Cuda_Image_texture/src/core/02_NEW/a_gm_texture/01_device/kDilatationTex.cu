#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"
#include "ifelse.cu.h"
#include "Indices.cu.h"

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * input  : tex2dGM 		(Image seuiller)
 * output : tabPixelsGM 	(Image ecran)
 */
__global__ void kDilatationTex(uchar* tabPixelsGM , hipTextureObject_t tex2dGM , uint w , uint h)
    {
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i;
    int j;
    int sum;
    while (s < WH)
	{
	Indices::toIJ(s, w, &i, &j);

	sum=0;

	// read
	    {
	    // ligne dessus
	    sum += tex2D < uchar > (tex2dGM, j - 1, i - 1);  // warning: orde i et j : tex2D(j,i) et non (i,j)
	    sum += tex2D < uchar > (tex2dGM, j, i - 1);
	    sum += tex2D < uchar > (tex2dGM, j + 1, i - 1);

	    // meme ligne, sauf moi meme
	    sum += tex2D < uchar > (tex2dGM, j - 1, i);
	    sum += tex2D < uchar > (tex2dGM, j + 1, i);

	    // ligne dessous
	    sum += tex2D < uchar > (tex2dGM, j - 1, i + 1);
	    sum += tex2D < uchar > (tex2dGM, j, i + 1);
	    sum += tex2D < uchar > (tex2dGM, j + 1, i + 1);
	    }

	// write
	tabPixelsGM[s] = ifelse(sum > 1, 255, 0);
	//tabPixelsGM[s]=tex2D < uchar > (tex2dGM, j, i); // debug

	// next
	s+=NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

