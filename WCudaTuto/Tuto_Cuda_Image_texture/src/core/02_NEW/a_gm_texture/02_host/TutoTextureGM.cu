#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "TutoTextureGM.h"
#include "GM.h"
#include "Hardware.h"

#include "OpencvTools_GPU.h"

using std::cout;
using std::cerr;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KSeuillage(uchar* tabPixelsSeuillerGM , uint wh , uint seuil0255);
extern __global__ void kDilatationTex(uchar* tabPixelsGM , hipTextureObject_t tex2dGM , uint w , uint h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

TutoTextureGM::TutoTextureGM(const Grid& gridDilatation , IteratorImageGray_I* ptrIteratorImage , uint seuil0255) :
	Animable_I<uchar>(gridDilatation, ptrIteratorImage->getW(), ptrIteratorImage->getH(), "Tuto_texture_GM_uchar"), //
//
	ptrIteratorImage(ptrIteratorImage), //
//
	gridSeuillage(bestGridSeuillage()), //
	gridDilatation(gridDilatation), //
	//
	wh(ptrIteratorImage->getWH()), seuil0255(seuil0255), //
//
	texture2D_GM(ptrIteratorImage->getW(), ptrIteratorImage->getH())
    {
    // Tools
    this->t = 0; // protected dans Animable

    // MM
	{
	this->sizeImage = sizeof(uchar) * wh;
	GM::malloc(&tabPixelSeuillerGM, sizeImage);
	}
    //video
	{
	this->tabPixelVideo = ptrIteratorImage->first();
	}
    }

TutoTextureGM::~TutoTextureGM()
    {
    GM::free(tabPixelSeuillerGM);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void TutoTextureGM::process(uchar* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    //                cpyHtoD                         seuillage                       lunette                dilatation
    // tabPixelVideo --------->  tabPixelsSeuillerGM  ----------> tabPixelsSeuillerGM --------> texture2D_GM---------> tabPixelGM
    //						                                                             read texture
    //               Etape 1                           Etape 2                        Etape 3                Etape 4

    // Etape 1
    GM::memcpyHToD(tabPixelSeuillerGM, tabPixelVideo, sizeImage);

    // Etape 2
    KSeuillage<<<gridSeuillage.dg,gridSeuillage.db>>>(tabPixelSeuillerGM , wh, seuil0255);

    // Etape 3
    texture2D_GM.see(tabPixelSeuillerGM);

    // Etape 4
    kDilatationTex<<<gridDilatation.dg,gridDilatation.db>>>(tabPixelsGM , texture2D_GM.texCuda , w , h);
    }

/**
 * Override
 * Call periodicly by the API
 */
void TutoTextureGM::animationStep()
    {
    t++;

    this->tabPixelVideo = ptrIteratorImage->next();
    }

/*-------------------------*\
 |*	private		    *|
 \*-------------------------*/

/**
 * static
 */
Grid TutoTextureGM::bestGridSeuillage()
    {
    const int MP = Hardware::getMPCount();
    const int CORE_MP = Hardware::getCoreCountMP();

    dim3 dg(MP * 2);
    dim3 db(CORE_MP * 7);
    Grid gridConstrastor(dg, db);

    return gridConstrastor;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

