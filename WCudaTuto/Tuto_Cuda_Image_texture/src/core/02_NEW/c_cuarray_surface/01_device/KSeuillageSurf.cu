#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"

#include "Calibreur.cu.h"
#include "Indices.cu.h"
#include "ifelse.cu.h"


/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/
/**
 * tabPixelGM   : read
 * surf2d       : write
 */
__global__ void KSeuillageSurf(hipSurfaceObject_t surf2d, uchar* tabPixelVideoGM , uint w , uint h , uchar seuil0255)
    {
    const int WH = w * h;
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    int s = TID;
    int i;
    int j;
    uchar pixelS;
    while (s < WH)
	{
	Indices::toIJ(s, w, &i, &j);

	// read
	pixelS = tabPixelVideoGM[s];

	// work
	pixelS = ifelse(pixelS < seuil0255, 0, 255);

	// write
	surf2Dwrite(pixelS, surf2d, j, i); // warning : (j,i) et non (j,i)

	// next
	s += NB_THREAD;
	}
    }


/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

