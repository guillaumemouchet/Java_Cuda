#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"

#include "Calibreur.cu.h"
#include "Indices.cu.h"
#include "ifelse.cu.h"
#include "Edge.cu.h"

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * surf2d 	: read
 * tabPixelGM   : write
 *
 * Note: fait par exemple disparaitre les points noirs dans les bandes blanches de la route (pour video autoroute)
 */
__global__ void KDilatationSurf(uchar* tabPixelGM , hipSurfaceObject_t surf2d , uint w , uint h)
    {
    const int WH = w * h;
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    // Edge edge(w, h, 1);

    int s = TID;
    int i;
    int j;
    uchar pixelS;
    int sum;
    while (s < WH)
	{
	Indices::toIJ(s, w, &i, &j);

	surf2Dread(&pixelS, surf2d, j, i); // warning (j,i) et non (i,j)

	//if (!edge.isInEdge(i, j)) // pas besoin avec hipBoundaryModeClamp ci-dessous
	    {
	    sum = 0;

	    // read
		{
		// ligne dessus
		sum += surf2Dread < uchar > (surf2d, j - 1, i - 1, hipBoundaryModeClamp);
		sum += surf2Dread < uchar > (surf2d, j, i - 1, hipBoundaryModeClamp);
		sum += surf2Dread < uchar > (surf2d, j + 1, i - 1, hipBoundaryModeClamp);

		// meme ligne, sans moi
		sum += surf2Dread < uchar > (surf2d, j - 1, i, hipBoundaryModeClamp);
		sum += surf2Dread < uchar > (surf2d, j + 1, i, hipBoundaryModeClamp);

		// ligne dessous
		sum += surf2Dread < uchar > (surf2d, j - 1, i + 1, hipBoundaryModeClamp);
		sum += surf2Dread < uchar > (surf2d, j, i + 1, hipBoundaryModeClamp);
		sum += surf2Dread < uchar > (surf2d, j + 1, i + 1, hipBoundaryModeClamp);
		}

	    // write
	    tabPixelGM[s] = ifelse(sum >= 1, (uchar)255, pixelS);
	    //tabPixelGM[s] = pixelS; // debug
	    }

	// next
	s += NB_THREAD;
	}
    }

// Note:
// 	on pourrait prendre un voisinage v8 avec une (ou plusieurs) aurole de plus, autour du pixels,
// 	pour faire disparaitre encore plus de zone noir dans la bande blanche, quitte a fire une errosion apres
//	Voir KDilatationN ci-dessous

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

