#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "TutoSurface.h"
#include "GM.h"
#include "Hardware.h"

#include "OpencvTools_GPU.h"

using std::cout;
using std::cerr;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KSeuillageSurf(hipSurfaceObject_t surf2d, uchar* tabPixelVideoGM , uint w , uint h , uchar seuil0255);
extern __global__ void KDilatationSurf(uchar* tabPixelsGM , hipSurfaceObject_t surf2d , uint w , uint h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

TutoSurface::TutoSurface(const Grid& gridDilatation , IteratorImageGray_I* ptrIteratorImage , uint seuil0255) :
	Animable_I<uchar>(gridDilatation, ptrIteratorImage->getW(), ptrIteratorImage->getH(), "Tuto_surface_uchar"), //
//
	ptrIteratorImage(ptrIteratorImage), //
//
	gridSeuillage(bestGridSeuillage()), //
	gridDilatation(gridDilatation), //
//
	wh(ptrIteratorImage->getWH()), seuil0255(seuil0255), //
//
	surface2D(ptrIteratorImage->getW(), ptrIteratorImage->getH(),CudaArrayType::SURFACE_LOAD_STORE) // SURFACE_LOAD_STORE DEFAULT_CUARRAY
    {
    // Tools
    this->t = 0; // protected dans Animable

    // MM
	{
	this->sizeImage = sizeof(uchar) * wh;
	GM::malloc(&tabPixelVideoGM, sizeImage);
	}
    //video
	{
	this->tabPixelVideo = ptrIteratorImage->first();
	}
    }

TutoSurface::~TutoSurface()
    {
    GM::free(tabPixelVideoGM);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void TutoSurface::process(uchar* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    //                cpyHtoD                    seuillage             dilatation
    // tabPixelVideo --------->  tabPixelVideoGM ----------> surface2D ---------> tabPixelGM
    //                                          write Surface          read Surface
    //                Etape 1                    Etape 2               Etape 3

    // Etape 1
    GM::memcpyHToD(tabPixelVideoGM, tabPixelVideo, sizeImage);

    // Etape 2
    KSeuillageSurf<<<gridSeuillage.dg,gridSeuillage.db>>>(surface2D.surfCuda , tabPixelVideoGM,w,h, seuil0255);

    // Etape 3
    KDilatationSurf<<<gridDilatation.dg,gridDilatation.db>>>(tabPixelsGM ,surface2D.surfCuda, w , h);
    }

/**
 * Override
 * Call periodicly by the API
 */
void TutoSurface::animationStep()
    {
    t++;

    this->tabPixelVideo = ptrIteratorImage->next();
    }

/*-------------------------*\
 |*	private		    *|
 \*-------------------------*/

/**
 * static
 */
Grid TutoSurface::bestGridSeuillage()
    {
    const int MP = Hardware::getMPCount();
    const int CORE_MP = Hardware::getCoreCountMP();

    dim3 dg(MP * 2);
    dim3 db(CORE_MP * 8);
    Grid gridConstrastor(dg, db);

    return gridConstrastor;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

