#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "TutoTextureCuarray.h"
#include "GM.h"
#include "Hardware.h"

#include "OpencvTools_GPU.h"

using std::cout;
using std::cerr;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KSeuillage(uchar* tabPixelVideoGM , uint wh , uint seuil0255);
extern __global__ void kDilatationTex(uchar* tabPixelsGM , hipTextureObject_t tex2dGM , uint w , uint h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

TutoTextureCuarray::TutoTextureCuarray(const Grid& gridDilatation , IteratorImageGray_I* ptrIteratorImage , uint seuil0255) :
	Animable_I<uchar>(gridDilatation, ptrIteratorImage->getW(), ptrIteratorImage->getH(), "Tuto_texture_uchar"), //
//
	ptrIteratorImage(ptrIteratorImage), //
//
	gridSeuillage(bestGridSeuillage()), //
	gridDilatation(gridDilatation), //
//
	wh(ptrIteratorImage->getWH()), seuil0255(seuil0255), //
//
	texture2DVideo(ptrIteratorImage->getW(), ptrIteratorImage->getH())
    {
    // Tools
    this->t = 0; // protected dans Animable

    // MM
	{
	this->sizeImage = sizeof(uchar) * wh;
	GM::malloc(&tabPixelVideoGM, sizeImage);
	}

    //video
	{
	this->tabPixelVideo = ptrIteratorImage->first();
	}
    }

TutoTextureCuarray::~TutoTextureCuarray()
    {
    GM::free(tabPixelVideoGM);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void TutoTextureCuarray::process(uchar* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    //                cpyHtoD                    seuillage                     cpyDtoD             dilatation
    // tabPixelVideo --------->  tabPixelVideoGM ----------> tabPixelVideoGM ----------> texture2D ---------> tabPixelGM
    //  								                          read Texture
    //		      Etape 1			   Etape 2             Etape 3                    Etape 4

    // Etape 1
    GM::memcpyHToD(tabPixelVideoGM, tabPixelVideo, sizeImage);

    // Etape 2
    KSeuillage<<<gridSeuillage.dg,gridSeuillage.db>>>(tabPixelVideoGM , wh, seuil0255);

    // Etape 3
    texture2DVideo.memcpyDtoD(tabPixelVideoGM);

    // Etape 4
    kDilatationTex<<<gridDilatation.dg,gridDilatation.db>>>(tabPixelsGM , texture2DVideo.texCuda, w , h);
    }

/**
 * Override
 * Call periodicly by the API
 */
void TutoTextureCuarray::animationStep()
    {
    t++;

    this->tabPixelVideo = ptrIteratorImage->next();
    }

/*-------------------------*\
 |*	private		    *|
 \*-------------------------*/

/**
 * static
 */
Grid TutoTextureCuarray::bestGridSeuillage()
    {
    const int MP = Hardware::getMPCount();
    const int CORE_MP = Hardware::getCoreCountMP();

    dim3 dg(MP * 8);
    dim3 db(CORE_MP * 2);
    Grid gridConstrastor(dg, db);

    return gridConstrastor;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

