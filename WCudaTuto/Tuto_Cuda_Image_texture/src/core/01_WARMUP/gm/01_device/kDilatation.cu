#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"

#include "ifelse.cu.h"
#include "Edge.cu.h"
#include "Indices.cu.h"
#include "SousImageIterator.cu.h"

/*----------------------------------------------------------------------*\
 |*			Declaration / private				*|
 \*---------------------------------------------------------------------*/

static __device__ void kDilatationV1(uchar* tabPixelsGM , uchar* tabPixelVideoSeuillerGM , uint w , uint h);
static __device__ void kDilatationV2(uchar* tabPixelsGM , uchar* tabPixelVideoSeuillerGM , uint w , uint h);
static __device__ void kDilatationV3(uchar* tabPixelsGM , uchar* tabPixelVideoSeuillerGM , uint w , uint h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void kDilatation(uchar* tabPixelsGM , uchar* tabPixelVideoSeuillerGM , uint w , uint h)
    {
   // kDilatationV1(tabPixelsGM, tabPixelVideoSeuillerGM, w, h);
  //  kDilatationV2(tabPixelsGM, tabPixelVideoSeuillerGM, w, h);
    kDilatationV3(tabPixelsGM, tabPixelVideoSeuillerGM, w, h);
    }

/*----------------------------------------------------------------------*\
 |*			private 					*|
 \*---------------------------------------------------------------------*/

/*-------------*\
 |*	v1	*|
 \*------------*/

__device__ void kDilatationV1(uchar* tabPixelsGM , uchar* tabPixelVideoSeuillerGM , uint w , uint h)
    {
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;

    Edge edge(w, h, 1); // debug bord=100

    int s = TID;
    int sum;
    int i;
    int j;
    while (s < WH)
	{
	Indices::toIJ(s, w, &i, &j);
	if (!edge.isInEdge(i, j)) // thread divergence
	    {
	    sum = 0;

	    // read
		{
		// ligne dessus
		sum += tabPixelVideoSeuillerGM[s - w - 1];
		sum += tabPixelVideoSeuillerGM[s - w];
		sum += tabPixelVideoSeuillerGM[s - w + 1];

		// meme ligne, sauf moi meme
		sum += tabPixelVideoSeuillerGM[s - 1];
		sum += tabPixelVideoSeuillerGM[s + 1];

		// ligne dessous
		sum += tabPixelVideoSeuillerGM[s + w - 1];
		sum += tabPixelVideoSeuillerGM[s + w];
		sum += tabPixelVideoSeuillerGM[s + w + 1];
		}

	    // write
	    tabPixelsGM[s] = ifelse(sum > 1, 255, 0);
	    }

	// next
	s += NB_THREAD;
	}
    }

/*-------------*\
 |*	v2	*|
 \*------------*/

/**
 * sans if
 * sans thead-divergence
 */
__device__ void kDilatationV2(uchar* tabPixelsGM , uchar* tabPixelVideoSeuillerGM , uint w , uint h)
    {
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    //	Difficulter:
    //
    //		(O1)	Partern entrelacement sur la zone centrale (sans if)
    //
    // Solution:
    //
    //		(S1) Step A: on fabrique s puis (i,j) de la petiteImage
    //	             Step B: on fabrique ss puis (i.j) de la grandeImage
    // Definition:
    //
    //		(D1) zoneCentrale=image-bord ou bord depend fu rayon du kernel de convolution
    //
    //		(D2) petiteImage= image de base, de taille de la zone centrale, avec le coin en haut a gauche au meme endroit que image de bas
    //
    //		(D3) petimeImageCentrale = petiteImage translater en i et j de rayon et rayon
    //
    //		(D4) Grande image = image de base wxh
    //
    //	Notes:
    //
    //		(N1) Le resultat de la convolution doit etre calculer que sur petimeImageCentrale
    //
    //		(N2) petiteImage et petimeImageCentrale sont de meme taille, mais pas eu meme endroit
    //

    const int BORD = 1; // debug 100, voir zone centrale
    const int W_INTERNE = w - 2 * BORD;
    const int H_INTERNE = h - 2 * BORD;
    const int WH_INTERNE = W_INTERNE * H_INTERNE;

    int sum;
    int s = TID;
    int i;
    int j;
    while (s < WH_INTERNE) // on parcours autant de pixels que contient petiteImage
	{
	/*--------------------------------------*\
	 |*	preparation indice sous image	*|
	 \*-------------------------------------*/

	// Step A: (i,j,s) petite image:
	Indices::toIJ(s, W_INTERNE, &i, &j); // on fabrique (i,j) de la petite image

	// Step B: (i,j,ss) grande image:
	// Step B.1: on fabrique le (i,j) correspondant de la grande image par translation dans la zone centrale
	i += BORD;
	j += BORD;
	// Step B.2: on fabrique ss de la grande image en partant de (i,j) translater
	int ss = (i * w) + j;

	// resumer:
	//	avant (i,j,s) petiteImage
	//	apres (i,j,ss) grandeImage

	/*--------------------------------------*\
	|*	travail avec indice sous image	*|
	 \*-------------------------------------*/

	// travai avec (i,j,ss)
	sum = 0;

	//read
	    {
	    // ligne dessus
	    sum += tabPixelVideoSeuillerGM[ss - w - 1];
	    sum += tabPixelVideoSeuillerGM[ss - w];
	    sum += tabPixelVideoSeuillerGM[ss - w + 1];

	    // meme ligne, sauf moi meme
	    sum += tabPixelVideoSeuillerGM[ss - 1];
	    sum += tabPixelVideoSeuillerGM[ss + 1];

	    // ligne dessous
	    sum += tabPixelVideoSeuillerGM[ss + w - 1];
	    sum += tabPixelVideoSeuillerGM[ss + w];
	    sum += tabPixelVideoSeuillerGM[ss + w + 1];
	    }

	// write
	tabPixelsGM[ss] = ifelse(sum > 1, 255, 0);
	//tabPixelsGM[ss] = tabPixelVideoSeuillerGM[ss]; // debug

	// next
	s += NB_THREAD;
	}
    }

/*-------------*\
 |*	v3	*|
 \*------------*/

/**
 * sans if
 * sans thead-divergence
 * idem v2, plus light (SousImageIterator contient l'approche ci-dessus de v2)
 */
__device__ void kDilatationV3(uchar* tabPixelsGM , uchar* tabPixelVideoSeuillerGM , uint w , uint h)
    {
    const int BORD = 1; // debug 100
    SousImageIterator it(w, h, BORD);

    int sum;
    int i;
    int j;
    int s;
    while (it.hasNext())
	{
	s = it.nextS(&i, &j);

	sum = 0;

	// read
	    {
	    // ligne dessus
	    sum += tabPixelVideoSeuillerGM[s - w - 1];
	    sum += tabPixelVideoSeuillerGM[s - w];
	    sum += tabPixelVideoSeuillerGM[s - w + 1];

	    // meme ligne, sauf moi meme
	    sum += tabPixelVideoSeuillerGM[s - 1];
	    sum += tabPixelVideoSeuillerGM[s + 1];

	    // ligne dessous
	    sum += tabPixelVideoSeuillerGM[s + w - 1];
	    sum += tabPixelVideoSeuillerGM[s + w];
	    sum += tabPixelVideoSeuillerGM[s + w + 1];
	    }

	// write
	tabPixelsGM[s] = ifelse(sum > 1, 255, 0);
	//tabPixelsGM[s] = tabPixelVideoSeuillerGM[s]; // ou 128 DEBUG
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

