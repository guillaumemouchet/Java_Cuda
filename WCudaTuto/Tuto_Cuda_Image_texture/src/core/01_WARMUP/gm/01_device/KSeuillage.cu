#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"
#include "ifelse.cu.h"


/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void KSeuillage(uchar* tabPixelVideoGM , uint wh , uint seuil0255)
    {
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    int s = TID;
    while (s < wh)
	{
	//v1
//	{
//	if ( tabPixelsGM[s] > seuil0255)
//	    {
//	    tabPixelsGM[s] = 255
//	    }
//	else
//	    {
//	    tabPixelsGM[s] = 0;
//	    }
//	}

	// v2
	tabPixelVideoGM[s] = ifelse(tabPixelVideoGM[s] > seuil0255,255,0);

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			 private					*|
 \*---------------------------------------------------------------------*/

/**
 * Supression thread divergence:
 *
 * 	if (t)
 * 		return a;
 * 	else
 * 		return b;
 *
 * qui est identique a a thread divergence de l'operateur ternaire
 *
 * 	return t?a:b
 */
//__device__ uchar ifelse(uchar t , uchar a , uchar b)
//    {
//    return (1 - t) * b + t * a;
//    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

