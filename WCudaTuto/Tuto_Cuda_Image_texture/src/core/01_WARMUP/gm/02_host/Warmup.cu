#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Warmup.h"
#include "GM.h"
#include "Hardware.h"

#include "OpencvTools_GPU.h"

using std::cout;
using std::cerr;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KSeuillage(uchar* tabPixelVideoGM , uint wh , uint seuil0255);
extern __global__ void kDilatation(uchar* tabPixelsGM , uchar* tabPixelVideoSeuillerGM , uint w , uint h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Warmup::Warmup(const Grid& gridDilatation , IteratorImageGray_I* ptrIteratorImage,uint seuil0255) :
	Animable_I<uchar>(gridDilatation, ptrIteratorImage->getW(), ptrIteratorImage->getH(), "Warmup_dilatation_uchar"), //
	//
	ptrIteratorImage(ptrIteratorImage),	//
	//
	gridSeuillage(bestGridSeuillage()), //
	gridDilatation(gridDilatation), //
		//
	wh(ptrIteratorImage->getWH()),
	//
	seuil0255(seuil0255)
    {
    // Tools
    this->t = 0; // protected dans Animable

    // MM
	{
	this-> sizeImage = sizeof(uchar) * wh;
	GM::malloc(&tabPixelSeuillerGM, sizeImage);
	}

    //video
	{
	this->tabPixelVideo = ptrIteratorImage->first();
	}
    }

Warmup::~Warmup()
    {
    GM::free (tabPixelSeuillerGM);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Warmup::process(uchar* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    //                cpyHtoD                         seuillage                      dilatation
    // tabPixelVideo --------->  tabPixelsSeuillerGM ----------> tabPixelsSeuillerGM ---------> tabPixelGM
    //		      Etape 1                         Etape 2                        Etape 3

    // Etape 1
    GM::memcpyHToD(tabPixelSeuillerGM, tabPixelVideo, sizeImage);

    // Etape 2
    KSeuillage<<<gridSeuillage.dg,gridSeuillage.db>>>(tabPixelSeuillerGM , wh,  seuil0255);

    // Etape 3
    kDilatation<<<gridDilatation.dg,gridDilatation.db>>>(tabPixelsGM , tabPixelSeuillerGM , w , h);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Warmup::animationStep()
    {
    t++;

    this->tabPixelVideo = ptrIteratorImage->next();
    }

/*-------------------------*\
 |*	private		    *|
 \*-------------------------*/


/**
 * static
 */
Grid Warmup::bestGridSeuillage()
    {
    const int MP = Hardware::getMPCount();
    const int CORE_MP = Hardware::getCoreCountMP();

    dim3 dg(MP * 1);
    dim3 db(1024);
    Grid gridConstrastor(dg, db);

    return gridConstrastor;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

