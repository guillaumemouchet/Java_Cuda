#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "cudas.h"
#include "VagueGrayMath.cu.h"

#include "Indices.cu.h"

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void vagueGray(uchar* tabPixelsGM , uint w , uint h , int t)
    {
    VagueGrayMath vagueGrayMath(w, h, t);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;

    int i; // in [0,h[
    int j; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &i, &j); 	// update (i, j)

	vagueGrayMath.colorIJ(&tabPixelsGM[s], i, j); 	// update ptrDevPixels[s]

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

