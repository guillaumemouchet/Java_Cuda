#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "cudas.h"

#include "DamierRGBAFloatMath.cu.h"
#include "Indices.cu.h"

#include "DomaineMath.h"


/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void damierRGBAFloat(float4* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , uint n , float t)
    {
    DamierRGBAFloatMath damierRGBAFloatMath(n, t);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;

    double x;
    double y;

    int i; // in [0,h[
    int j; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &i, &j); // update (i, j)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(i, j, &x, &y); //  (i,j) -> (x,y)

	damierRGBAFloatMath.colorXY(&tabPixelsGM[s], x, y); // update ptrDevPixels[s]

	s += NB_THREAD;
	}

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

