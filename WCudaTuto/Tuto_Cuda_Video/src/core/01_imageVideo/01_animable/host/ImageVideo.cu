#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "ImageVideo.h"
#include "GM.h"

using std::cout;
using std::cerr;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void kToGray(uchar4* tabPixelGM,uint wh);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

ImageVideo::ImageVideo(const Grid& grid , uint w , uint h , string nameVideo , int nbImage) :
	Animable_I<uchar4>(grid, w, h, "Video_RGBA_uchar4"), //
	videoProviderRGB(nameVideo, nbImage)
    {
    // Tools
    this->t = 0; // protected dans Animable
    this->sizeImage = sizeof(uchar4) * w * h;

    //video
	{
	bool isOk = videoProviderRGB.loadALL();
	assert(isOk);
	assert(videoProviderRGB.getW() == w && videoProviderRGB.getH() == h);

	animationStep();
	}
    }

ImageVideo::~ImageVideo()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void ImageVideo::process(uchar4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    GM::memcpyHToD(tabPixelsGM, ptrTabPixelVideo, sizeImage);

    kToGray<<<dg,db>>>(tabPixelsGM, w * h);
    }

/**
 * Override
 * Call periodicly by the API
 */
void ImageVideo::animationStep()
    {
    t++;

    this->ptrTabPixelVideo = videoProviderRGB.getImage();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

