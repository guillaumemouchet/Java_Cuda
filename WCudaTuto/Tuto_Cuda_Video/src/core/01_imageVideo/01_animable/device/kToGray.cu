#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "cudas.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		private			*|
 \*-------------------------------------*/

static __device__ void toGray(uchar4* pixel);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void kToGray(uchar4* tabPixelsGM , uint wh)
    {
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    int s = TID;
    while (s < wh)
	{
	toGray(&tabPixelsGM[s]); //update color

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void toGray(uchar4* ptrPixelColor)
    {
    const float POIDS=1.0f/3.0f;

    uchar r = ptrPixelColor->x;
    uchar g = ptrPixelColor->y;
    uchar b = ptrPixelColor->z;

    uchar levelGris = r * POIDS + g * POIDS + b * POIDS;

    ptrPixelColor->x = levelGris;
    ptrPixelColor->y = levelGris;
    ptrPixelColor->z = levelGris;

    ptrPixelColor->w = 255; // opaque
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

