#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "ImageFile.h"
#include "GM.h"

using std::cout;
using std::cerr;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void kToGray(uchar4* tabPixelGM,uint wh);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

ImageFile::ImageFile(const Grid& grid , Image& image) :
	Animable_I<uchar4>(grid, image.w(), image.h(), "ImageFile_RGBA_uchar4"), //
	image(image)
    {
    // Tools
    this->t = 0; // protected dans Animable
    this->sizeImage = sizeof(uchar4) * w * h;

    this->ptrTabPixelImage = image.uchar4_RGBA();
    }

ImageFile::~ImageFile()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void ImageFile::process(uchar4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    GM::memcpyHToD(tabPixelsGM, ptrTabPixelImage, sizeImage);

kToGray<<<dg,db>>>(tabPixelsGM, w * h);
}

/**
 * Override
 * Call periodicly by the API
 */
void ImageFile::animationStep()
{
t++;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

