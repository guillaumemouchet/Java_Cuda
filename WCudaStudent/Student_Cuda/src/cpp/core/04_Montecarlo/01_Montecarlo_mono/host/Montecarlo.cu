#include "hip/hip_runtime.h"
#include "Montecarlo.h"

#include <iostream>
#include <assert.h>
#include <typeinfo>
#include <math.h>

#include "GM.h"
#include "Hardware.h"
#include "Stream.h"

using std::cout;
using std::endl;
using std::string;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void createGenerator(hiprandState* tabGeneratorGM,int deviceId);

extern __global__ void montecarlo(hiprandState* tabGeneratorGM, entier nbDarByThread, entier* ptrNbDarUnderGM, float h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(const Grid& grid , entier nbDarTotalAsk , double* ptrPiHat , float h , bool isVerbose) :
	RunnableGPU(grid, title(nbDarTotalAsk, h), isVerbose), // classe parente
	//
	nbDarTotalAsk(nbDarTotalAsk), //
	ptrPiHat(ptrPiHat), //
	h(h) //

    {
    const entier NB_THREAD = grid.threadCounts(); //directement en entier pour éviter un cast

    // Math : Calculer le nombre de flechette effectivment tirer!
	{
	this->nbDarByThread = -1; //TODO Montecarlo
	this->nbDarTotalEffective = -1; // TODO Montecarlo

	assert(nbDarTotalAsk >= grid.threadCounts());
	assert(nbDarByThread > 0); // si =0, trop de threads et pas assez de dar!
	}

    // MM
	{
	this->sizeNbDarUnderGM = -1; // [octet]

	// TODO Montecarlo MM NbDarUnderGM (pas oublier de mettre a zero, avec un malloc0 par exemple)

	this->sizeSM = -1; // [octet]
	}

    // init : lancer le kernel de creation des generators
	{
	size_t sizeTabDevGeneratorGM = -1; // TODO Montecarlo

	// TODO Montecarlo MM pou  tabDevGeneratorGM

	// TODO Montecarlo lancer le kernel createGenerator
	int deviceId = Hardware::getDeviceId();

	}
    }

Montecarlo::~Montecarlo(void)
    {
//MM (device free)
	{
	// TODO Montecarlo MM
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * Lancer kernel
 * Recuperer resultat coter host
 * Finaliser le calcul de pi
 */
void Montecarlo::run()
    {
    // Version :  mono pure
	{
	// TODO Montecarlo

	}

    // Version : compatible with multiGPU, stream-version
	{
	// rien pour version mono pure
	// puis des la version MontecarloMulti-stream
	// commenter la version ci-dessus, et utiliser :
	//	- kernel_async
	//	- DtoH_async
	//	- Stream::synchronize(0);
	// Valider cette npuvelle version, d'abord en reexecutamt une Montecarlo(Mono)
	// TODO Montecarlo MontecarloMulti-stream
	}

    // calcule de Math
    // TODO Montecarlo
    }

/*----------------*\
 |*	get        *|
 \*---------------*/

entier Montecarlo::getNbDarTotalEffective()
    {
    return nbDarTotalEffective;
    }

entier Montecarlo::getNbDarUnderCurve()
    {
    return nbDarUnderCurve;
    }

double Montecarlo::getInputGO()
    {
    return (nbDarTotalEffective / (double)1024 / (double)1024 / (double)1024) * sizeof(float) * 2;
    }

/*----------------*\
 |*  private       *|
 \*---------------*/

/**
 * static
 */
string Montecarlo::title(entier nbDarTotalAsk , float h)
    {
    return "Montecarlo-" + entierToString() + "-" + to_string(nbDarTotalAsk) + "-h" + to_string((int)h);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
