#include "hip/hip_runtime.h"
#include "AddVector.h"

#include <iostream>
#include <assert.h>

#include "Kernel.h"
#include "GM.h"
#include "Bandwidth.h"
#include "VectorTools.h"
#include "Stream.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addVector(int* ptrDevV1, int* ptrDevV2, int* ptrDevW,int n,int sOffset=0);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVector::AddVector(const Grid& grid , int* ptrV1 , int* ptrV2 , int* ptrW , int n , bool isVerbose) :
	RunnableGPU(grid, title() + "-" + to_string(n), isVerbose), // classe parente
	//
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n)
    {
    this->sizeVector = n * sizeof(int); // octet

    // MM (malloc Device)
	{
	GM::malloc(&ptrDevV1, sizeVector);
	GM::malloc(&ptrDevV2, sizeVector);
	GM::malloc(&ptrDevW, sizeVector);
	}

    }

AddVector::~AddVector(void)
    {
    //MM (device free)
	{
	GM::free(ptrDevV1);
	GM::free(ptrDevV2);
	GM::free(ptrDevW);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * override
 */
void AddVector::run()
    {
    // MM (copy Host->Device)
	{
	Bandwidth bandwidth(sizeVector * 2, "[" + title() + "] : Host -> GM :");

	GM::memcpyHToD(ptrDevV1, ptrV1, sizeVector);
	GM::memcpyHToD(ptrDevV2, ptrV2, sizeVector);

	if (isVerbose) // dans Runable ou RunnableGPU
	    {
	    cout << bandwidth << endl;
	    }
	}

    addVector<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n);	// assynchrone

    // MM (Device -> Host)
	{
	Bandwidth bandwidth(sizeVector, "[" + title() + "] : GM -> Host :");

	GM::memcpyDToH(ptrW, ptrDevW, sizeVector);

	if (isVerbose)
	    {
	    cout << bandwidth << endl;
	    }
	}
    }

/**
 * override
 */
double AddVector::getInputGO()
    {
    return ((long)2 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/**
 * override
 */
double AddVector::getOutputGO()
    {
    return ((long)1 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

string AddVector::title()
    {
    if (VectorTools::isDMA())
	{
	return "Addvector-DMA-int";
	}
    else
	{
	return "Addvector-int";
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
