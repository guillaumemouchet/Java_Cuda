#include "hip/hip_runtime.h"
#include "AddVectorTristream.h"

#include <iostream>
#include <assert.h>

#include "Kernel.h"
#include "GM.h"
#include "Bandwidth.h"
#include "VectorTools.h"
#include "Stream.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/


extern __global__ void addVector(int* ptrDevV1 , int* ptrDevV2 , int* ptrDevW , int n,int sliceIndex=0);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVectorTristream::AddVectorTristream(const Grid& grid , int* ptrV1 , int* ptrV2 , int* ptrW , int n , int nbSlice , bool isVerbose) :
	RunnableGPU(grid, title(nbSlice) + "-" + to_string(n), isVerbose), // classe parente
	//
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n), //
	nbSlice(nbSlice)
    {
    assert(n % nbSlice == 0); // on suppose que chaque slice a la meme taille
    assert(nbSlice >= 3);

    this->sizeVector = n * sizeof(int); // octet
    this->SIZE_SLICE = -1; // TODO addVector en octet
    this->N_BY_SLICE = -1; // TODO addVector nb case by slice

    // MM (malloc Device)
	{
	GM::malloc0(&ptrDevV1, sizeVector);
	GM::malloc0(&ptrDevV2, sizeVector);
	GM::malloc0(&ptrDevW, sizeVector);
	}

    // Stream
	{
	// TODO addVector, see attribute in .h
	assert(false); // to remove once coded
	}
    }

AddVectorTristream::~AddVectorTristream(void)
    {
    //MM (device free)
	{
	GM::free(ptrDevV1);
	GM::free(ptrDevV2);
	GM::free(ptrDevW);
	}

    // Stream
	{
	// TODO addVector, see attribute in .h
	assert(false); // to remove once coded
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * Conseils :
 *
 * 	(C1) Commencer par les warmup pour un nombre de slice petit et fixe, puis passer seulement ensuite a la generalisation.
 * 	(C2) Pour les warmup, prenez une taille n de vecteur petit (pour pouvoir afficher le resultat)
 * 	     Aller dans VectorTools::n(); // activez la ligne debug provisoirement, et dans nDebug() prenez une valeur peitite et divisible par votre nombre se slice
 * 	(C3) Travailler au début avec mainUse.cpp (utiliser le blog debug fournit avec flag de verbosite a false avec !)
 */
void AddVectorTristream::run()
    {
    // Warmup
	{
	run3Slice();
	//run4Slice();
	//run5Slice();
	}

    //runGeneric(); // TODO addVector a activer une fois le warup valider

    // Synchronize
	{
	// TODO addVector
	}
    }

/**
 * Warmup : 3 slice
 */
void AddVectorTristream::run3Slice()
    {
    // Warning : use private methode:
    //			- void copyHtoD(int sliceIndex , hipStream_t stream)
    //			- void copyDtoH(int sliceIndex , hipStream_t stream)
    //			- kernelSlice(int sliceIndex , hipStream_t stream)
    // voir en bas de fichier

    // step1 :
	{
	// TODO addVector see schema in pdf
	}

    // step2 :
	{
	// TODO addVector see schema in pdf
	}

    // partie centrale 3 stream en parallel
	{
	// step3 : (1 seul step pour 3 slice et 3 stream)
	    {
	    // TODO addVector see schema in pdf
	    }
	}

    // step 4 :
	{
	// TODO addVector see schema in pdf
	}

    // step 5 :
	{
	// TODO addVector see schema in pdf
	}
    }

/**
 * Warmup : 4 slice
 */
void AddVectorTristream::run4Slice()
    {
    // Warning : use private methode:
    //			- void copyHtoD(int sliceIndex , hipStream_t stream)
    //			- void copyDtoH(int sliceIndex , hipStream_t stream)
    //			- kernelSlice(int sliceIndex , hipStream_t stream)

    // partie Init
	{
	// step1
	    {
	    // TODO addVector see schema in pdf
	    }

	// step2
	    {
	    // TODO addVector see schema in pdf
	    }
	}

    // partie centrale 3 stream en parallel
	{
	// step3
	    {
	    // TODO addVector see schema in pdf
	    }

	// step4
	    {
	    // TODO addVector see schema in pdf
	    }
	}

    // partie Finale
	{
	// step 4
	    {
	    // TODO addVector see schema in pdf
	    }

	//step 5
	    {
	    // TODO addVector see schema in pdf
	    }
	}
    }

/**
 * Warmup : 5 slice
 */
void AddVectorTristream::run5Slice()
    {
    // Warning : use private methode:
    //			- void copyHtoD(int sliceIndex , hipStream_t stream)
    //			- void copyDtoH(int sliceIndex , hipStream_t stream)
    //			- kernelSlice(int sliceIndex , hipStream_t stream)

    // partie Init
	{
	// step1
	    {
	    // TODO addVector see schema in pdf
	    }

	// step2
	    {
	    // TODO addVector see schema in pdf
	    }
	}

    // partie centrale 3 stream en parallel
	{
	// step3
	    {
	    // TODO addVector see schema in pdf
	    }

	// step4
	    {
	    // TODO addVector see schema in pdf
	    }

	// step 5
	    {
	    // TODO addVector see schema in pdf
	    }
	}

    // partie finale
	{
	const int INDEX_LAST = -1;        // TODO 		// un peu de genericiter
	const int INDEX_BEFORE_LAST = INDEX_LAST - 1; 	// un peu de genericiter

	// before last
	    {
	    // TODO addVector see schema in pdf
	    }

	// last
	    {
	    // TODO addVector see schema in pdf
	    }
	}
    }

void AddVectorTristream::runGeneric()
    {
    // Warning : use private methode:
    //			- void copyHtoD(int sliceIndex , hipStream_t stream)
    //			- void copyDtoH(int sliceIndex , hipStream_t stream)
    //			- kernelSlice(int sliceIndex , hipStream_t stream)

    // partie Init
	{
	// step1
	    {
	    // TODO addVector see schema in pdf
	    }

	// step2
	    {
	    // TODO addVector see schema in pdf
	    }
	}

    // 6 variable utile pour les permutations
    hipStream_t streamA = stream0; // hipStream_t est un int
    hipStream_t streamB = stream2;
    hipStream_t streamC = stream1;

    hipStream_t streamA_old = stream0;
    hipStream_t streamB_old = stream2;
    hipStream_t streamC_old = stream1;

    // partie centrale 3 stream en parallel
	{
	// TODO addVector see schema in pdf
	}

    // partie finale
	{
	const int INDEX_LAST = nbSlice - 1;
	const int INDEX_BEFORE_LAST = INDEX_LAST - 1;

	// before last
	    {
	    // TODO addVector see schema in pdf
	    }

	// last
	    {
	    // TODO addVector see schema in pdf
	    }
	}
    }

/**
 * override
 */
double AddVectorTristream::getInputGO()
    {
    return ((long)2 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/**
 * override
 */
double AddVectorTristream::getOutputGO()
    {
    return ((long)1 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * copyHtoD: la slice sliceIndex pour
 * 	v1
 * 	v2
 * ou
 * 	sliceIndex in [0,nbSlice[
 */
void AddVectorTristream::copyHtoD(int sliceIndex , hipStream_t stream)
    {
    const int OFFSET_SLICE = sliceIndex * N_BY_SLICE;

    // TODO addVector
    // Copier sur le device la slice de v1 correspondand a sliceIndex  sur la stream demander
    // idem pour v2
    }

/**
 * copyDtoH: la slice sliceIndex pour
 * 	w
 * ou
 * 	sliceIndex in [0,nbSlice[
 */
void AddVectorTristream::copyDtoH(int sliceIndex , hipStream_t stream)
    {
    const int OFFSET_SLICE = sliceIndex * N_BY_SLICE;

    // TODO addVector
    // Copier sur le host la slice de w correspondand à sliceIndex  sur la stream demander
    }

/**
 * lance le kernel de calcul pour la slice sliceIndex
 * ou
 * 	sliceIndex in [0,nbSlice[
 */
void AddVectorTristream::kernelSlice(int sliceIndex , hipStream_t stream)
    {
    const int OFFSET_SLICE = sliceIndex * N_BY_SLICE;

    // TODO addVector
    // appeler le kernel sur la slice correspondant a sliceIndex
    }

string AddVectorTristream::title(int nbSlice)
    {
    if (VectorTools::isDMA())
	{
	return "Addvector-tristream-slice" + to_string(nbSlice) + "-DMA-int";
	}
    else
	{
	return "Addvector-tristream-slice" + to_string(nbSlice) + "-DMA-int";
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
