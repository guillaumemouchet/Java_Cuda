#include "hip/hip_runtime.h"
#include "AddVectorTristream.h"

#include <assert.h>
#include <GM.h>
#include <GM_MemoryManagement.h>
#include <Stream.h>
#include <iostream>

#include "../../00_tools/VectorTools.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addVector(int* ptrDevV1 , int* ptrDevV2 , int* ptrDevW , int n,int sliceIndex=0);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVectorTristream::AddVectorTristream(const Grid& grid , int* ptrV1 , int* ptrV2 , int* ptrW , int n , int nbSlice , bool isVerbose) :
	RunnableGPU(grid, title(nbSlice) + "-" + to_string(n), isVerbose), // classe parente
	//
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n), //
	nbSlice(nbSlice)
    {
    assert(n % nbSlice == 0); // on suppose que chaque slice a la meme taille
    assert(nbSlice >= 3);

    this->sizeVector = n * sizeof(int); // octet
    this->SIZE_SLICE = sizeVector / nbSlice; // TODO addVector en octet
    this->N_BY_SLICE = n / nbSlice; // TODO addVector nb case by slice

    // MM (malloc Device)
	{
	GM::malloc0(&ptrDevV1, sizeVector);
	GM::malloc0(&ptrDevV2, sizeVector);
	GM::malloc0(&ptrDevW, sizeVector);
	}

    // Stream
	{
	// TODO addVector, see attribute in .h
	Stream::create(&stream0);
	Stream::create(&stream1);
	Stream::create(&stream2);
	}
    }

AddVectorTristream::~AddVectorTristream(void)
    {
    //MM (device free)
	{
	GM::free(ptrDevV1);
	GM::free(ptrDevV2);
	GM::free(ptrDevW);
	}

    // Stream
	{
	// TODO addVector, see attribute in .h
	Stream::destroy(stream0);
	Stream::destroy(stream1);
	Stream::destroy(stream2);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * Conseils :
 *
 * 	(C1) Commencer par les warmup pour un nombre de slice petit et fixe, puis passer seulement ensuite a la generalisation.
 * 	(C2) Pour les warmup, prenez une taille n de vecteur petit (pour pouvoir afficher le resultat)
 * 	     Aller dans VectorTools::n(); // activez la ligne debug provisoirement, et dans nDebug() prenez une valeur peitite et divisible par votre nombre se slice
 * 	(C3) Travailler au début avec mainUse.cpp (utiliser le blog debug fournit avec flag de verbosite a false avec !)
 */
void AddVectorTristream::run()
    {
    // Warmup
	{
	//run3Slice();
	//run4Slice();
	//run5Slice();
	}

    runGeneric();
    // Synchronize
	{
	Stream::synchronize(stream0);
	Stream::synchronize(stream1);
	Stream::synchronize(stream2);

	}
    }

/**
 * Warmup : 3 slice
 */
void AddVectorTristream::run3Slice()
    {
    // Warning : use private methode:
    //			- void copyHtoD(int sliceIndex , hipStream_t stream)
    //			- void copyDtoH(int sliceIndex , hipStream_t stream)
    //			- kernelSlice(int sliceIndex , hipStream_t stream)
    // voir en bas de fichier

    // step1 :
	{
	copyHtoD(0, stream0);
	}

    // step2 :
	{
	copyHtoD(1, stream1);
	kernelSlice(0, stream0);

	}

// partie centrale 3 stream en parallel

    // step3 : (1 seul step pour 3 slice et 3 stream)
	{
	copyHtoD(2, stream2);
	kernelSlice(1, stream1);
	copyDtoH(0, stream0);

	}

    // step 4 :
	{
	kernelSlice(2, stream2);
	copyDtoH(1, stream1);

	}

// step 5 :
	{
	copyDtoH(2, stream2);
	}
    }

/**
 * Warmup : 4 slice
 */
void AddVectorTristream::run4Slice()
    {

// Warning : use private methode:
//			- void copyHtoD(int sliceIndex , hipStream_t stream)
//			- void copyDtoH(int sliceIndex , hipStream_t stream)
//			- kernelSlice(int sliceIndex , hipStream_t stream)

// partie Init
	{
	// step1
	    {
	    copyHtoD(0, stream0);
	    }

	// step2
	    {
	    copyHtoD(1, stream1);
	    kernelSlice(0, stream0);
	    }
	}

// partie centrale 3 stream en parallel
	{
	// step3
	    {
	    copyHtoD(2, stream2);
	    kernelSlice(1, stream1);
	    copyDtoH(0, stream0);
	    }

	// step4
	    {
	    copyHtoD(3, stream0);
	    kernelSlice(2, stream2);
	    copyDtoH(1, stream1);
	    }
	}

// partie Finale
	{
	// step 4
	    {
	    kernelSlice(3, stream0);
	    copyDtoH(2, stream2);
	    }

	//step 5
	    {
	    copyDtoH(3, stream0);

	    }
	}
    }

/**
 * Warmup : 5 slice
 */
void AddVectorTristream::run5Slice()
    {
// Warning : use private methode:
//			- void copyHtoD(int sliceIndex , hipStream_t stream)
//			- void copyDtoH(int sliceIndex , hipStream_t stream)
//			- kernelSlice(int sliceIndex , hipStream_t stream)

// partie Init
	{
	// step1
	    {
	    copyHtoD(0, stream0);

	    }

	// step2
	    {
	    copyHtoD(1, stream1);
	    kernelSlice(0, stream0);
	    }
	}

// partie centrale 3 stream en parallel
	{
	// step3
	    {
	    copyHtoD(2, stream2);
	    kernelSlice(1, stream1);
	    copyDtoH(0, stream0);
	    }

	// step4
	    {
	    copyHtoD(3, stream0);
	    kernelSlice(2, stream2);
	    copyDtoH(1, stream1);
	    }

	// step 5
	    {
	    copyHtoD(4, stream1);
	    kernelSlice(3, stream0);
	    copyDtoH(2, stream2);
	    }
	}

// partie finale
	{
	const int INDEX_LAST = -1;        // TODO 		// un peu de genericiter
	const int INDEX_BEFORE_LAST = INDEX_LAST - 1; 	// un peu de genericiter

	// before last
	    {
	    kernelSlice(INDEX_LAST, stream1);
	    copyDtoH(INDEX_BEFORE_LAST, stream0);
	    }

	// last
	    {
	    copyDtoH(INDEX_LAST, stream1);

	    }
	}
    }

void AddVectorTristream::runGeneric()
    {
// Warning : use private methode:
//			- void copyHtoD(int sliceIndex , hipStream_t stream)
//			- void copyDtoH(int sliceIndex , hipStream_t stream)
//			- kernelSlice(int sliceIndex , hipStream_t stream)

// partie Init
	{
	// step1
	    {
	    copyHtoD(0, stream0);
	    }

	// step2
	    {
	    copyHtoD(1, stream1);
	    kernelSlice(0, stream0);
	    }
	}

// 6 variable utile pour les permutations
    hipStream_t streamA = stream0; // hipStream_t est un int
    hipStream_t streamB = stream2;
    hipStream_t streamC = stream1;

    hipStream_t streamA_old = stream0;
    hipStream_t streamB_old = stream2;
    hipStream_t streamC_old = stream1;

// partie centrale 3 stream en parallel
	{
	for (int i = 2; i < nbSlice; i++)
	    {
	    copyHtoD(i, streamB);
	    kernelSlice(i - 1, streamC);
	    copyDtoH(i - 2, streamA);
	    /*
	     *  copyHtoD(2, stream2); ==B
	     *	kernelSlice(1, stream1); ==C
	     *	copyDtoH(0, stream0); ==A
	     */
	    // Reset old stream
	    streamA_old = streamA;
	    streamB_old = streamB;
	    streamC_old = streamC;

	    //set streams to new values in a circular way A->C->B->A
	    streamA = streamC_old;
	    streamB = streamA_old;
	    streamC = streamB_old;

	    }
	}

// partie finale
	{
	const int INDEX_LAST = nbSlice - 1;
	const int INDEX_BEFORE_LAST = INDEX_LAST - 1;

	// before last
	    {
	    kernelSlice(INDEX_LAST, streamB_old);
	    copyDtoH(INDEX_BEFORE_LAST, streamC_old);
	    }

	// last
	    {
	    copyDtoH(INDEX_LAST, streamB_old);

	    }
	}
    }

/**
 * override
 */
double AddVectorTristream::getInputGO()
    {
    return ((long)2 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/**
 * override
 */
double AddVectorTristream::getOutputGO()
    {
    return ((long)1 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * copyHtoD: la slice sliceIndex pour
 * 	v1
 * 	v2
 * ou
 * 	sliceIndex in [0,nbSlice[
 */
void AddVectorTristream::copyHtoD(int sliceIndex , hipStream_t stream)
    {
    const int OFFSET_SLICE = sliceIndex * N_BY_SLICE;

// addVector
// Copier sur le device la slice de v1 correspondand a sliceIndex  sur la stream demander
// idem pour v2
    GM::memcpyAsyncHToD(ptrDevV1 + OFFSET_SLICE, ptrV1 + OFFSET_SLICE, SIZE_SLICE, stream);
    GM::memcpyAsyncHToD(ptrDevV2 + OFFSET_SLICE, ptrV2 + OFFSET_SLICE, SIZE_SLICE, stream);
    }

/**
 * copyDtoH: la slice sliceIndex pour
 * 	w
 * ou
 * 	sliceIndex in [0,nbSlice[
 */
void AddVectorTristream::copyDtoH(int sliceIndex , hipStream_t stream)
    {
    const int OFFSET_SLICE = sliceIndex * N_BY_SLICE;

// addVector
// Copier sur le host la slice de w correspondand à sliceIndex  sur la stream demander
    GM::memcpyAsyncDToH(ptrW + OFFSET_SLICE, ptrDevW + OFFSET_SLICE, SIZE_SLICE, stream);
    }

/**
 * lance le kernel de calcul pour la slice sliceIndex
 * ou
 * 	sliceIndex in [0,nbSlice[
 */
void AddVectorTristream::kernelSlice(int sliceIndex , hipStream_t stream)
    {
    const int OFFSET_SLICE = sliceIndex * N_BY_SLICE;

// addVector
// appeler le kernel sur la slice correspondant a sliceIndex
    addVector<<<dg,db,0,stream>>>(ptrDevV1 + OFFSET_SLICE, ptrDevV2 + OFFSET_SLICE, ptrDevW + OFFSET_SLICE, N_BY_SLICE, sliceIndex);
}

string AddVectorTristream::title(int nbSlice)
{
    if (VectorTools::isDMA())
	{
	return "Addvector-tristream-slice" + to_string(nbSlice) + "-DMA-int";
	}
    else
	{
	return "Addvector-tristream-slice" + to_string(nbSlice) + "-DMA-int";
	}
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
