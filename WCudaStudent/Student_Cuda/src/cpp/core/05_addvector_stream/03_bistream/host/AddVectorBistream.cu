#include "hip/hip_runtime.h"
#include "AddVectorBistream.h"

#include <iostream>
#include <assert.h>

#include "Kernel.h"
#include "GM.h"
#include "Bandwidth.h"
#include "VectorTools.h"
#include "Stream.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/


extern __global__ void addVector(int* ptrDevV1, int* ptrDevV2, int* ptrDevW,int n,int sliceIndex=0);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVectorBistream::AddVectorBistream(const Grid& grid , int* ptrV1 , int* ptrV2 , int* ptrW , int n , bool isVerbose) :
	RunnableGPU(grid, title() + "-" + to_string(n), isVerbose), // classe parente
	//
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n)
    {
    assert(n % 2 == 0); // on suppose que les 2 slice ont la meme taille

    this->sizeVector = n * sizeof(int); // octet

    // MM (malloc Device)
	{
	GM::malloc0(&ptrDevV1, sizeVector);
	GM::malloc0(&ptrDevV2, sizeVector);
	GM::malloc0(&ptrDevW, sizeVector);
	}

    // Stream
	{
	// TODO addVector, see attribute in .h
	assert(false); // to remove once coded
	}
    }

AddVectorBistream::~AddVectorBistream(void)
    {
    //MM (device free)
	{
	GM::free(ptrDevV1);
	GM::free(ptrDevV2);
	GM::free(ptrDevW);
	}

    // Stream
	{
	// TODO addVector
	assert(false); // to remove once coded
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * override
 */
void AddVectorBistream::run()
    {
    const int MIDLE = n >> 1; // n/2
    const size_t MIDLE_SIZE = sizeVector >> 1; // sizeVector/2

    // Step 1:
	{
	// TODO addVector see schema in pdf
	// stream0 : copieHtoD  slice0

	Stream::synchronize(stream0);
	}

    // Step 2:
	{
	// TODO addVector see schema in pdf
	// stream1 : copieHtoD  slice1
	// stream0 : kernel     slice0
	// Warning : il faut lancer le kernel sur une slice!, pas sur tout le veteur!
	//	(W1)	Attention a la dimension a donner
	//	(W2)	Attention au sliceIndex,  ie 0
	}

    // Step 3:
	{
	// TODO addVector see schema in pdf
	// stream0 : copieDtoH   slice0
	// stream1 : kernel      slice1
	// Warning : il faut lancer le kernel sur la slice1, pas sur tout le veteur!
	//	(W1)	Attention a la dimension a donner, ie le nombre de case de la slice, ie MIDLE
	//	(W2) 	Attention au sliceIndex, ie 1
	//	(W3)	Attention a  l'arithmetic des pointeurs!
	//		on veut travailler non pas sur:
	//				ptrDev1
	//				ptrDev2
	//				ptrDevW
	//		mais sur :
	//				ptrDev1+MIDLE
	//				ptrDev2+MIDLE
	//				ptrDevW+MIDLE
	}

    // Step 4:
	{
	// TODO addVector see schema in pdf
	// stream1 : copieDtoH      slice1
	}

    // Synchronize
	{
	// v1 : best
	    {
	    Stream::synchronize(stream0);
	    Stream::synchronize(stream1);
	    }

	// v2 :bad (au cas ou d'autre TP sont lancer en meme temps
	//Stream::synchronize();
	}
    }

/**
 * override
 */
double AddVectorBistream::getInputGO()
    {
    return ((long)2 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/**
 * override
 */
double AddVectorBistream::getOutputGO()
    {
    return ((long)1 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

string AddVectorBistream::title()
    {
    if (VectorTools::isDMA())
	{
	return "Addvector-bistream-DMA-int";
	}
    else
	{
	return "Addvector-bistream-int";
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
