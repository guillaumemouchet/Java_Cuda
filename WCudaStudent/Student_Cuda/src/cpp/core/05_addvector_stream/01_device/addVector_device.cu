#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Limits.h"

/*----------------------------------------------------------------------*\
 |*			tools 					*|
 \*---------------------------------------------------------------------*/

__device__ int inc(int t)
    {
    float a = t;

    //int inc = round(cosf(a) * cosf(a) + sinf(a) * sinf(a)); // 1
    int inc = __float2int_rn(cosf(a) * cosf(a) + sinf(a) * sinf(a)); // 1

    return inc;
    }

/**
 * But:
 *
 * 	Faire perdre du temps, sans changer u
 *
 * Raison:
 *
 * 	Pas besoîn d'un gpu pour additionner deux vecteurs
 * 	Le transfert sur le pci-express prend plus de temps que n'en a besoin le cpu pour faire l'addition.
 *
 * Note:
 *
 * 	On simule ici un calcul plus compliquer et long.
 * 	Mais le bu du du TP, n'est pas de se compliquer la vie avec un algorithme compliqué coté device
 * 	mais de se concentrer sur les streams cote host
 *
 * Piege :
 *
 * 	Le compilateur nvcc detecte les codes morts et les enleves.
 * 	Si la fonction loseTime n'emploie pas une variable utiliser dans le resultat final
 * 	la fonction loseTime risque d'etre supprimer en tant que code mort.
 * 	On utiliser ici u qui est le resultat de l'addition, pour est certain que loseTome ne soit pas supprimer code code mort.
 */
__device__ int loseTime(int u)
    {
    // Plus le GPU est performant plus il faut prendre grand
    const int N = 400; // chercher speed up de 1.9 voir 2

    // decrementer
	{
	int t = 0;
	while (t < N)
	    {
	    t++;
	    u = u - inc(t);
	    }
	}

    // incrementer
	{
	int t = 0;
	while (t < N)
	    {
	    t++;
	    u = u + inc(t);
	    }
	}

    return u;
    }

/**
 * C'est quoi ptrDevV1?
 *
 * 	Version baseline : Tout le vecteur
 *
 * 	Version bistream : Seulement une partie du vecteur, soit la slice0, soit la slice1, et cette slice a deux fois moins de case que le vecteur complet
 * 		     	   n est la taille de la slice
 *
 * 	Version tristream : Seulement une partie du vecteur, une des slices

 */
__device__ void process(int* ptrDevV1 , int* ptrDevV2 , int* ptrDevW , int s , int n,int sliceIndex)
    {
    int sGlobal = s + (sliceIndex+n);

    //1) additioner la composante s:	us=v1(s)+v2(s)
    //2) appeler loseTime sur us, loseTime ne modifie pas us
    //3) le resultat final est loseTime(us), ie les cases s additionner, mais avec une perte de temps
    //
    //	 		resultatS=loseTime(v1(s)+v2(s))

    // TODO

    // TIP : pour debuguer, mettez au début:
    //
    //		ptrDevW[s]=sGlobal;
    //
    // 	     Si vous avez un bug, ca permetra de savoir si ca vient du host ou du device
    }

/*----------------------------------------------------------------------*\
 |*			main	 					*|
 \*---------------------------------------------------------------------*/

/**
 * output : void required, car kernel is asynchrone
 * pattern entrelacement
 */
__global__ void addVector(int* ptrDevV1 , int* ptrDevV2 , int* ptrDevW , int n , int sliceIndex = 0)
    {
    const int NB_THREAD = Thread1D::nbThread();
    const int TID = Thread1D::tid();

   // TODO addVector
    // entrelacement et call process ci-dessus
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

