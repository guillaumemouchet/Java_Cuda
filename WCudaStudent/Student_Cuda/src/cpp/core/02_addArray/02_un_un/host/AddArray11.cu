#include "hip/hip_runtime.h"
#include "AddArray11.h"

#include <iostream>
#include <assert.h>

#include "Kernel.h"
#include "GM.h"
#include "VectorTools.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addArray11(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddArray11::AddArray11(const Grid& grid , float* ptrV1 , float* ptrV2 , float* ptrW , int n) :
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n), //
	dg(grid.dg), //
	db(grid.db)
    {
    this->sizeVector = n*sizeof(float); // TODO addArray11 // octet

    // MM (malloc Device)
	{
	GM::malloc(&ptrDevV1,sizeVector);
	GM::malloc(&ptrDevV2,sizeVector);

	GM::malloc(&ptrDevW,sizeVector);

	}
    }

AddArray11::~AddArray11(void)
    {
    //MM (device free)
	{
	GM::free(ptrDevV1);
	GM::free(ptrDevV2);

	GM::free(ptrDevW);

	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * override
 */
void AddArray11::run()
    {
    // MM (copy Host->Device)
	{
	GM::memcpyHToD(ptrDevV1, ptrV1, sizeVector);
	GM::memcpyHToD(ptrDevV2, ptrV2, sizeVector);

	}

    assert(dg.x * dg.y * dg.z * db.x * db.y * db.z == n);
    // TODO addArray11 call kernel

    addArray11<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n);
    // MM (Device -> Host)
	{
	GM::memcpyDToH(ptrW, ptrDevW, sizeVector);
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
