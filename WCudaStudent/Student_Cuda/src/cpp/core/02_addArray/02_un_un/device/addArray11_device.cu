#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * <pre>
 * Output :
 *
 * 		void required, car kernel is asynchrone!
 *
 * Contrainte:
 * 		pattern 1<-->1
 * 		On associe un thread a chaque case du tableau
 * </pre>
 */
__global__ void addArray11(float* ptrDevV1 , float* ptrDevV2 , float* ptrDevW , int n)
    {
    const int TID = Thread2D::tid();

   ptrDevW[n] = ptrDevV1[n] + ptrDevV2[n];
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

