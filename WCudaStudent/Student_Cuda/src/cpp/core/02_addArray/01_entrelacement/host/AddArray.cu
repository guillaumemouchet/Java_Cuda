#include "hip/hip_runtime.h"
#include "AddArray.h"

#include <assert.h>
#include <GM.h>
#include <GM_MemoryManagement.h>
#include <iostream>

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addArray(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddArray::AddArray(const Grid& grid , float* ptrV1 , float* ptrV2 , float* ptrW , int n) :
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n), // this.n = n;
	dg(grid.dg), //
	db(grid.db)
    {
    this->sizeVector = n * sizeof(float); // octet

    // MM (malloc Device)
	{
	GM::malloc(&ptrDevV1, sizeVector);
	GM::malloc(&ptrDevV2, sizeVector);
	GM::malloc(&ptrDevW, sizeVector);

	}

    }

AddArray::~AddArray(void)
    {
    //MM (device free)
	{
	GM::free(ptrDevV1);
	GM::free(ptrDevV2);
	GM::free(ptrDevW);

	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * override
 */
void AddArray::run()
    {
    // MM (copy Host->Device)
	{
	GM::memcpyHToD(ptrDevV1, ptrV1, sizeVector); //ptrDevV1 = ptrV1 <-
	GM::memcpyHToD(ptrDevV2, ptrV2, sizeVector);

	}

    // call kernel // assynchrone
    addArray<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n);
    //Kernel::synchronize();// inutile

    // MM (Device -> Host)
	{
	// MM barier de synchronisation implicite
	GM::memcpyDToH(ptrW, ptrDevW, sizeVector);
	}
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*----------------------------------------------------------------------*/
