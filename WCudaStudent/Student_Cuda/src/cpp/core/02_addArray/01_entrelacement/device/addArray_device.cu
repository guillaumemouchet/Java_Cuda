#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * output : void required, car kernel is asynchrone
 * pattern entrelacement
 */
__global__ void addArray(float* ptrDevV1 , float* ptrDevV2 , float* ptrDevW , int n)
    {
    const int NB_THREAD = Thread2D::nbThread();
    const int TID = Thread2D::tid();

    // Debug, facultatif
//    if (TID == 0)
//	{
//	printf("Coucou from device tid = %d", TID);
//	}
    int s = TID;
    while(s<NB_THREAD)
	{
	ptrDevW[s] = ptrDevV1[s] + ptrDevV2[s];
	s +=NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

