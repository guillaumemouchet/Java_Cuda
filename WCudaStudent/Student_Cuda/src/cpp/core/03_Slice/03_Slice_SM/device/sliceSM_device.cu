#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "ReductionAdd.cu.h"

#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(float* tabSM,int nbSlice);
static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void sliceSM(int nbSlice , float* ptrPiHatGM)
    {
    // TODO SliceSM

    // Declarer tabSM
    // ReductionIntraThread
    // Reduction de tabSM (use tools ReductionAdd)

    // __syncthreads(); necessaire? ou?
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * remplit la sm
 */
__device__ void reductionIntraThread(float* tabSM , int nbSlice)
    {
    // TODO SliceSM

    // Warning: Il faut employer TID et TID_LOCAL
    }

__device__ float f(float x)
    {
    // TODO SliceSM
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

