#include "hip/hip_runtime.h"
#include "SliceSM.h"

#include <iostream>
#include <assert.h>

#include "GM.h"
#include "SM.h"

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void sliceSM(int nbSlice,float* ptrPiHatGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

SliceSM::SliceSM(const Grid& grid , int nbSlice , double* ptrPiHat , bool isVerbose) :
	RunnableGPU(grid, "SliceSM-" + to_string(nbSlice), isVerbose), // classe parente
	//
	ptrPiHat(ptrPiHat), //
	nbSlice(nbSlice) //
    {
    this->ptrPiHatGM = NULL;
    this->sizeSM = -1; //TODO SliceSM

    // MM
	{
	// TODO SliceSM (pas oublier de mettre a zero, avec mallocfloat0 par exemple)

	// Tip:		Il y a une methode dedier pour malloquer un float cote device et l'initialiser a zero
	//
	//		   GM::mallocfloat0(&ptrGM);
	}
    }

SliceSM::~SliceSM(void)
    {
    // TODO SliceSM
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void SliceSM::run()
    {
    // Etape 1 : lancer le kernel
    // Etape 2 : recuperer le resultat coter host (par exemple avec memcpyDToH_float)
    // Etape 3 : finaliser le calcul de PI

    // TODO SliceSM
    }

// BruteForce:
//
//	Observation:

//		(O2)
//			La taille des tableau promu en SM doit etre une puissance de 2, pour pouvoir lancer les ecrasements
//			Or tabSM a autant de cases qu il y a de threads èar blocks
//			Et on aimerait bien respecter l'heuristique
//
//				(H2) 	nbThreadByBlock % nbCoreByMp = 0
//
//	Question :
//
//			Peut on respecter la contrainte "puissance de 2" et l'heuristique (H2)?
//
//	Reponse:
//			Tout depend du nombre de core par MP, mais c'est souvent 32 ou 64 ou 128
//			ie dans les 3 cas une puissance de 2, youpie
//
//			Comme il y a max 1024 threads par block, il n'y a pas beaucoup de possibilites
//
//				1024 512 256 128 64 32
//
//			Lors du forcebrute, il faut donc adapter les grilles a cette contrainte.
//			Regarde a cet effet la methode sliceSM dans mainBruteforce.cpp
//			et plus particulierement la methode bruteforceReduction utilisee.
//

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
