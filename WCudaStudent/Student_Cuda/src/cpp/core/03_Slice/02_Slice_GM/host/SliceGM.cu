#include "hip/hip_runtime.h"
#include "SliceGM.h"

#include <iostream>
#include <assert.h>

#include "GM.h"
#include "Maths.h"
#include "Hardware.h"
#include "Kernel.h"

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void reductionIntraThreadGM(float* tabGM,int nbSlice);
extern __global__ void ecrasementGM(float* tabGM, int moitier);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

SliceGM::SliceGM(Grid grid , int nbSlice , double* ptrPiHat , bool isVerbose) :
	RunnableGPU(grid, "SliceGM-" + to_string(nbSlice), isVerbose), // classe parente
//
	nbSlice(nbSlice), //
	ptrPiHat(ptrPiHat) //
    {
    // ntabGM
	{
	this->nTabGM = -1; // TODO SliceGM

	}

    // MM
	{
	this->sizeTabGM = -1; //  TODO SliceGM // [octet]

	}
    }

SliceGM::~SliceGM(void)
    {
    //MM (device free)
	{
	//TODO SliceGM
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * <pre>
 * Idea globale
 *
 *	Etape 0 : Promotion d'un tableau en GM (MemoryManagement MM)
 * 	Etape 1 : Reduction intra-thread dans un tableau promu en GM
 * 	Etape 2 : Reduction du tableau en GM par ecrasement hierarchique 2 à 2
 * 		  On lance les kernels d'ecrasement depuis le host (chef d'orchestre)
 * 	Etape 4 : Copy du resultat coter host
 * 	Etape 5 : Destruction GM
 * </pre>
 */
void SliceGM::run()
    {
    //TODO SliceGM // call the kernel (asynchrone)

    reductionGM();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Etape 1 : Lancement des kernels d'ecrasement depuis le host, dans une boucle,
 * Etape 2 : recuperer le resultat coter host
 * Etape 3 : finaliser le calcule de PI
 */
void SliceGM::reductionGM()
    {
    int midle = nTabGM >> 1; // nTabGM/2;

    // TODO SliceGM

    // Warning:		Utiliser une autre grille que celle heriter de la classe parente dg, db
    // 			Votre grid ici doit avoir une taille speciale!
    // 			N'utiliser donc pas les vraibales dg et db de la super classe

    // Tip:		Il y a une methode dedier pour ramener un float cote host
    //
    //				float resultat;
    //				GM::memcpyDtoH_float(&resultat,ptrResultGM);
    }

// BruteForce:
//
//	Observation:
//
//		(O1) 	Il faut bruteforcer uniquement le dg dg du kernel de reduction intraThread.
//			La grid pour les ecrasement est elle imposer par la taille du tableau en GM a reduire.
//
//		(O2)
//			La taille du tableau promu en GM doit etre une puissance de 2, pour pouvoir lancer les ecrasements
//			Or tabGM a autant de cases qu il y a de threads totals
//			Et on aimerait bien respecter les deux heuristiques
//
//				(H2)	nbBlock % nbMp = 0
//				(H1) 	nbThreadByBlock % nbCoreByMp = 0
//
//	Question :
//
//			Peut on respecter la conrainte "puissance de 2" et les heuristics?
//
//	TIP:
//			Dans le bruteforce on peut specifier deux iterateurs, qui se baladent sur dg et db
//			Il faut que cette grille respecte la contraite (prioriter) et si on pas de chance, pas les heuristiques
//
//			Regarder l'implementation speciale de SliceGM dans mainBrutefore.cpp.
//			On fait une propre grille dedier aux proprietes de l' exercice
//

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
