#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Thread2D.cu.h>

/*----------------------------------------------------------------------*\
 |*            Declaration                     *|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*        Private            *|
 \*-------------------------------------*/

static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*            Implementation                     *|
 \*---------------------------------------------------------------------*/

/**
 * <pre>
 * Chaque thread effecteur une reduction intrathread avec le patern d'entrelacement,
 * puis stocke son r?sultat dans SA case dans tabGM
 *
 * tabGM est un tableau promu, qui a autant de case que de thread
 * </pre>
 */
__global__ void reductionIntraThreadGM(float* tabGM , int nbSlice)
    {
    const double DX = 1 / (double)nbSlice;
    const int NB_THREADS = Thread2D::nbThread();
    const int TID = Thread2D::tid();
    int s = TID;
    double sumThread = 0;

    while (s < nbSlice)
	{
	sumThread += f(s * DX);

	s += NB_THREADS;
	}
    tabGM[TID] = sumThread;
    }

/**
 * <pre>
 * Effectue la reduction de tabGM cote device, par ecrasement 2 ? 2 successif.
 * Ce kernel d ecrasement est appeler depuis le host dans une boucle, avec le bon nombre de thread
 *
 * Hypothese : |tabGM| est une puissance de 2
 *
 * Output: le resultat de la reduction est tans tabGM[0]
 * </pre>
 */
__global__ void ecrasementGM(float* tabGM , int middle)
    {
    const int TID = Thread2D::tid();
    tabGM[TID] = tabGM[TID] + tabGM[TID + middle];
    }

/*--------------------------------------*\
 |*        Private            *|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    return 4.0f / (1 + x * x);
    }
