#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * <pre>
 * Chaque thread effecteur une reduction intrathread avec le patern d'entrelacement,
 * puis stocke son résultat dans SA case dans tabGM
 *
 * tabGM est un tableau promu, qui a autant de case que de thread
 * </pre>
 */
__global__ void reductionIntraThreadGM(float* tabGM , int nbSlice)
    {
   // TODO SliceGM (idem SliceGMHOST) pour cette partie
    }

/**
 * <pre>
 * Effectue la reduction de tabGM cote device, par ecrasement 2 à 2 successif.
 * Ce kernel d ecrasement est appeler depuis le host dans une boucle, avec le bon nombre de thread
 *
 * Hypothese : |tabGM| est une puissance de 2
 *
 * Output: le resultat de la reduction est tans tabGM[0]
 * </pre>
 */
__global__ void ecrasementGM(float* tabGM , int middle)
    {
    // TODO SliceGM
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    // TODO SliceGM
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

