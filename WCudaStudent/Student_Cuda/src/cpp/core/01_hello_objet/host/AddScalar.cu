#include "hip/hip_runtime.h"
#include "AddScalar.h"

#include <iostream>
#include <assert.h>
#include "GM.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addScalar(float a, float b, float* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddScalar::AddScalar(float a , float b , float* ptrSum) :
	a(a),//
	b(b), //
	ptrSum(ptrSum)
    {
    this->sizeFloat = sizeof(float); // [octet]

    // MM
	{
	GM::malloc(&ptrSumGM, sizeFloat);
	}
    }

AddScalar::~AddScalar(void)
    {
    //MM
	{
	GM::free(ptrSumGM);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void AddScalar::run()
    {
    // Grid : specifier le nombre de thread
    dim3 dg(1, 1, 1);
    dim3 db(1, 1, 1); // contrainte produit <=1024
    assert(dg.x * dg.y * dg.z * db.x * db.y * db.z == 1);// 1 seul thread suffit

    addScalar<<<dg,db>>>(a, b, ptrSumGM); // assynchrone

    //Device::synchronize(); // inutile

    // MM (Device -> Host)
	{
	GM::memcpyDToH(ptrSum, ptrSumGM, sizeFloat); // MM = barriere synchronisation implicite
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
