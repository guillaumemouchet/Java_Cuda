#include "hip/hip_runtime.h"
#include "cudas.h"

#include <stdio.h>


/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * Hyp : 1 seul thread, ie dg(1,1,1) et db (1,1,1)
 */
__global__ void addScalar(float a , float b , float* ptrSumGM)
    {
    *ptrSumGM = a + b;

    // debug (1 seule thread ici)
    printf("\n[Hello : Device side : addScalar object] %.0f + %.0f = %.0f", a, b, *ptrSumGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

