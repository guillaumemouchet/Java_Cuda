#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include <stdio.h>
#include <assert.h>
#include "cudas.h"
#include "GM.h"
#include "Kernel.h"

using std::cout;
using std::endl;

/**
 * http://www.icl.utes3/docs/cuda.htmk.edu/~mgatl
 */

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ static void addScalar(int a , int b , int* ptrDevSum);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Host			*|
 \*-------------------------------------*/

__host__ bool addScalar_procedurale() // __host__ facultatif
    {
    int a = 10;
    int b = 1;
    int sum;

    int* ptrSum = &sum;	// on host (CPU)
    int* ptrDevSum; 	// on device (GPU)

    // MM (memory managment)
    size_t size = sizeof(int); // [octet]
    GM::malloc(&ptrDevSum, size); // Device memory allocation (*)

    // Grid : Specifier number thread : ici 1 thread au total !
    dim3 dg(1, 1, 1);
    dim3 db = dim3(1, 1, 1); // syntaxe alternative

    addScalar<<<dg,db>>>(a,b,ptrDevSum); // assynchrone, call le kernel GPU addScalar

    Kernel::synchronize(); // inutile

    // MM (memory management)
	{
	GM::memcpyDToH(ptrSum, ptrDevSum, size);   // Device -> Host, MM = barrier de synchronisation
	GM::free(ptrDevSum); //  free memory create in (*)
	}

    cout << "\n[Hello : Host   side : addScalar procedurale] " << a << " + " << b << " = " << sum << endl;
    assert(sum == a + b);
    bool isOk = (sum == a + b);

    return isOk;
    }

/*--------------------------------------*\
 |*		Device			*|
 \*-------------------------------------*/

/**
 * Hyp : 1 seul thread, ie dg(1,1,1) et db (1,1,1)
 */
__global__ void addScalar(int a , int b , int* ptrDevSum)
    {
    *ptrDevSum = a + b; // Hyp: 1 seul thread (pas besoin de plus, pour additioner 2 nombre)

    // debug (1 seule thread ici)
    printf("[Hello : Device side : addScalar procedurale] %d + %d = %d", a, b, *ptrDevSum);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

