#include "hip/hip_runtime.h"
#include "Raytracing.h"

#include <iostream>

#include "MM.h"
#include "GM.h"
#include <assert.h>
#include "SphereCreator.h"
#include "nbSphere.h"
#include "Bandwidth.h"
#include "ChronoFactory.h"

using std::cout;
using std::endl;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres);
__global__ void kernelRaytacingSM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres);
__global__ void kernelRaytacingCM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres);
__global__ void kernelRaytacingCM2SM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres);

__host__ void uploadToCM(Sphere* tabSpheres , int nbSpheres);

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

static string titre(const MemoryType& memoryType);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

Raytracing::Raytracing(const Grid& grid , uint w , uint h , float dt , bool isVerbose , int nbSpheres , const MemoryType& memoryType) :
	Animable_I<uchar4>(grid, w, h, titre(memoryType), isVerbose), // super classe
	memoryType(memoryType), //
	dt(dt), //
	nbSpheres(nbSpheres)
    {
    // Tools
    this->sizeSpheres = nbSpheres * sizeof(Sphere);
    this->t = 0;

    // spheres
	{
	Chrono* ptrChrono = ChronoFactory::create();
	SphereCreator sphereCreator(nbSpheres, w, h);
	ptrChrono->stop();
	if (isVerbose)
	    {
	    cout << "\n[SphereCreator] : " << *ptrChrono << endl;
	    }
	delete ptrChrono;

	uploadToDevice(sphereCreator.getTabSphere());
	} // SphereCreator depiler, donc detruit, tabSphere cote host detruit!
    }

Raytracing::~Raytracing()
    {
    // TODO Raytracing GPU MemoryManagement free
    switch (memoryType)
	{
    case GM:
	{
	assert(false); // TODO  Raytracing  GM
	// ??

	break;
	}
    case CM:
	{
	// Indication: rien a detruire pour la CM
	// Note : la duree de vie pour la CM est une duree processus.

	break;
	}
    case SM:
	{
	// Indication: rien a detruire pour la SM
	// Note : la duree de vie pour la SM est une duree de vie de kernel, seulement!

	break;
	}
    case CM2SM:
	{
	// Indication: rien a detruire pour la SM et CM

	break;
	}
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Raytracing::process(uchar4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    // TODO Raytracing GPU lauch kernel

    switch (memoryType)
	{
    case GM:
	{
	assert(false);	    // TODO Raytracing GM delete once implement
	// Call the kernel kernelRaytacingGM (prototype line 20, about)
	break;
	}
    case CM:
	{
	assert(false);	     // TODO Raytracing CM to delete once implement
	// Call the kernel kernelRaytacingCM (prototype line 20, about)
	break;
	}
    case SM:
	{
	assert(false);	     // TODO Raytracing SM to delete once implement
	// Call the kernel kernelRaytacingSM (prototype line 20, about)
	break;
	}
    case CM2SM:
	{
	assert(false);	     // TODO Raytracing CM_2_SM to delete once implement
	// Call the kernel kernelRaytacingSMbyCM (prototype line 20, about)
	break;
	}
	}
    }

/**
 * Override
 */
void Raytracing::animationStep()
    {
    t += dt;
    }

/**
 * Override
 */
double Raytracing::getInputGO()
    {
    return NB_SPHERE * sizeof(Sphere) / ((double)1024 * (double)1024 * (double)1024);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

void Raytracing::uploadToDevice(Sphere* tabSpheres)
    {
    Bandwidth bandwidth(sizeSpheres, "\n" + titre(memoryType) + " : Host -> Device :");

    switch (memoryType)
	{
    case GM:
	{
	// TODO Raytracing GM uploadToDevice
	// But : copier les spheres en GM
	// MM pour la GM ( malloc et memcpy)
	// Utiliser la classe GM
	// Regarder bien les attributs de la classe avant dans le .h
	assert(false);	   // to be removed once implemented

	break;
	}
    case CM:
	{
	// TODO Raytracing CM uploadToDevice
	// But : copier les spheres en CM
	// Utiliser la methode importer uploadToCM et pister là!
	assert(false);	    // to be removed once implemented

	break;
	}
    case SM:
	{
	// TODO Raytracing SM uploadToDevice

	// Indication:
	//		Coter device, on copie GM to SM
	//		Il faut donc d'abord copier les spheres sur le device!
	//		Le code est donc le meme que GM
	assert(false);	   // to be removed once implemented

	break;
	}
    case CM2SM:
	{
	// Indication:
	//		Coter device, on copie CM to SM
	//		Il faut donc d'abord copier les spheres en CM
	//		Le code est donc le meme que CM
	assert(false);		    // TODO uploadToDevice CM2SM ://to be removed once implemented
	// ??
	break;
	}
	}

    if (isVerbose)
	{
	cout << endl << bandwidth << endl;
	}
    }

string titre(const MemoryType& memoryType)
    {
    switch (memoryType)
	{
    case GM:
	return "Raytracing-GM-uchar4";

    case CM:
	return "Raytracing-CM-uchar4";

    case SM:
	return "Raytracing-GM2SM-uchar4";

    case CM2SM:
	return "Raytracing-CM2SM-uchar4";

    default:
	{
	assert(false);
	return "ERROR";
	}
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

