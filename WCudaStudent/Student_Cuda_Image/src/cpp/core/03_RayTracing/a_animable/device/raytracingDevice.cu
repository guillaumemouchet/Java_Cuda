#include "hip/hip_runtime.h"
#include <assert.h>

#include "Thread2D.cu.h"
#include "cudas.h"
#include "Indices.cu.h"

#include "Sphere.h"
#include "nbSphere.h"

#include "raytracingCM.cu.h"
#include "RaytracingMath.cu.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void work(uchar4* tabPixelsGM, uint w, uint h, float t, Sphere* tabSpheresDev, int nbSpheres);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres)
    {
    // TODO Raytracing GPU GM

    // Indications :
    //		(I1) Call methode work with good input
    // 		(I2) work contain the algo
    //		(I3) The algo is the same with the GM,CM,SM of TP Ractracing
    }

__global__ void kernelRaytacingSM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres)
    {
    // TODO Raytracing GPU SM

    // Indications :
    //		(I1) Copier les sphere de GM to SM
    // 		(I2) Call work with good input
    //		(I3) Implementer une methode copyGMtoSM
    }

__global__ void kernelRaytacingCM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres)
    {
    // TODO Raytracing GPU CM

    // Indications :
    //		(I1) call work with good input
    // 		(I2) TAB_SPHERES_CM est une variable globale a ce fichier! (Voir le debut de ce fichier, include rayTracingCM.h)
    }

__global__ void kernelRaytacingCM2SM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres)
    {
    // TODO Raytracing GPU SM_BY_CM

    // Indications :
    //		(I1) Copier les sphere de CM to SM
    // 		(I2) Call work with good input
    //		(I3) Implementer une methode copyCMtoSM
    //		     ou : Tip : renommer copyGMtoSM en copyDevToSM et utiliser copyDevToSM avec les bons inputs
    }

/*--------------------------------------*\
 |*		private			*|
 \*-------------------------------------*/

/**
 * Methode commune au 3 kernel ci-dessus.
 * Ici on ne sait pas si derriere tabSpheresDev, c'est
 * 	- de la GM?
 * 	- de la SM?
 * 	- de la CM?
 * Pas d'importance, c'est un pointeur et on travail avec!
 */
__device__ void work(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresDev , int nbSpheres)
    {
    // TODO Raytracing work device side

    // create RaytracingMath
    // entrelacement
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

