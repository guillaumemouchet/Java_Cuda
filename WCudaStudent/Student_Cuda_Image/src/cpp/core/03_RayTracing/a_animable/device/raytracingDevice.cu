#include "hip/hip_runtime.h"
#include <assert.h>

#include "Thread2D.cu.h"
#include "cudas.h"
#include "Indices.cu.h"

#include "Sphere.h"
#include "nbSphere.h"
#include "GM.h"


#include "raytracingCM.cu.h"
#include "RaytracingMath.cu.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void work(uchar4* tabPixelsGM, uint w, uint h, float t, Sphere* tabSpheresDev, int nbSpheres);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres)
    {
    // TODO Raytracing GPU GM

    // Indications :
    //		(I1) Call methode work with good input
    // 		(I2) work contain the algo
    //		(I3) The algo is the same with the GM,CM,SM of TP Ractracing
    work(tabPixelsGM,w,h,t,tabSpheresGM, nbSpheres);
    }

__global__ void kernelRaytacingSM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres)
    {
    // TODO Raytracing GPU SM
    extern __shared__ Sphere tabSM[];

    const int NB_THREADS = Thread2D::nbThreadLocal();
    const int TID_LOCAL = Thread2D::tidLocalBlock();

    int s = TID_LOCAL;

    while (s < nbSpheres)
       {
	tabSM[s] = tabSpheresGM[s];

   	s += NB_THREADS;
       }
    __syncthreads();

    work(tabPixelsGM,w,h,t,tabSM, nbSpheres);

    // Indications :
    //		(I1) Copier les sphere de GM to SM
    // 		(I2) Call work with good input
    //		(I3) Implementer une methode copyGMtoSM
    }

__global__ void kernelRaytacingCM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres)
    {
    // TODO Raytracing GPU CM

    // Indications :
    //		(I1) call work with good input
    // 		(I2) TAB_SPHERES_CM est une variable globale a ce fichier! (Voir le debut de ce fichier, include rayTracingCM.h)
    work(tabPixelsGM,w,h,t,TAB_SPHERES_CM, nbSpheres);
    }

__global__ void kernelRaytacingCM2SM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres)
    {
    // TODO Raytracing GPU SM_BY_CM
    extern __shared__ Sphere tabSM[];

    const int NB_THREADS = Thread2D::nbThreadLocal();
    const int TID_LOCAL = Thread2D::tidLocalBlock();

    int s = TID_LOCAL;

    while (s < nbSpheres)
       {
	tabSM[s] = TAB_SPHERES_CM[s];

	s += NB_THREADS;
       }
    __syncthreads();

    work(tabPixelsGM,w,h,t,tabSM, nbSpheres);

    // Indications :
    //		(I1) Copier les sphere de CM to SM
    // 		(I2) Call work with good input
    //		(I3) Implementer une methode copyCMtoSM
    //		     ou : Tip : renommer copyGMtoSM en copyDevToSM et utiliser copyDevToSM avec les bons inputs
    }

/*--------------------------------------*\
 |*		private			*|
 \*-------------------------------------*/

/**
 * Methode commune au 3 kernel ci-dessus.
 * Ici on ne sait pas si derriere tabSpheresDev, c'est
 * 	- de la GM?
 * 	- de la SM?
 * 	- de la CM?
 * Pas d'importance, c'est un pointeur et on travail avec!
 */
__device__ void work(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresDev , int nbSpheres)
    {
    // TODO Raytracing work device side
    // create RaytracingMath
    RaytracingMath raytracingMath(tabSpheresDev, nbSpheres, t);
    // entrelacement
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i = 0;
    int j = 0;
    while(s< WH)
	{
	Indices::toIJ(s,w,&i,&j);
	raytracingMath.colorIJ(&tabPixelsGM[s],i, j);
	s+=NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

