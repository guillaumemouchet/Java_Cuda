#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "cudas.h"

#include "real.h"
#include "MandelbrotMath.cu.h"

#include "DomaineMath.h"
#include "Indices.cu.h"


/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void mandelbrot(uchar4* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , int n)
    {
    MandelbrotMath mandelbrotMath(n);

    // TODO Mandelbrot GPU :
    //
    // entrelacement
    // s -> (i,j) -> (x,y)
    // appeler colorXY

    double x;
    double y;
    //domaineMath.toXY(i, j, &x, &y); // x et y doivent etre en double! Caster ensuite en real lors du passage à colorXY

    // Probleme : Choix a faire pour le type de (x,y) :
    //
    //			- Travailler en float pour plus de performance?
    //			- Travailler en double pour pouvoir zoomer plus loin dans le fractale?
    //
    // Solution : Travailler avec le typedef real definit dans "real.h"
    //		  Specifier dans "real.h" si vous souhaitez travailler en float ou en double
    //	          Ainsi on passe facilement et proprement d'une version float a une version double
    //
    // Note : domaineMath.toXY travaille lui toujours en double pour vous permettre les deux possibilites ci-dessus
    //
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

