#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "cudas.h"

#include "real.h"
#include "MandelbrotMath.cu.h"

#include "DomaineMath.h"
#include "Indices.cu.h"


/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void mandelbrot(uchar4* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , int n)
    {
    MandelbrotMath mandelbrotMath(n);

    // TODO Mandelbrot GPU :
    //
    // entrelacement
    // s -> (i,j) -> (x,y)
    // appeler colorXY

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i = 0;
    int j = 0;

    double x;
    double y;

    while (s < WH)
    	{
    	Indices::toIJ(s, w, &i, &j);
    	domaineMath.toXY(i, j, &x, &y);
    	mandelbrotMath.colorXY(&tabPixelsGM[s], (real)x, (real)y);
    	s += NB_THREAD;
    	}

    //domaineMath.toXY(i, j, &x, &y); // x et y doivent etre en double! Caster ensuite en real lors du passage à colorXY

    // Probleme : Choix a faire pour le type de (x,y) :
    //
    //			- Travailler en float pour plus de performance?
    //			- Travailler en double pour pouvoir zoomer plus loin dans le fractale?
    //
    // Solution : Travailler avec le typedef real definit dans "real.h"
    //		  Specifier dans "real.h" si vous souhaitez travailler en float ou en double
    //	          Ainsi on passe facilement et proprement d'une version float a une version double
    //
    // Note : domaineMath.toXY travaille lui toujours en double pour vous permettre les deux possibilites ci-dessus
    //
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
