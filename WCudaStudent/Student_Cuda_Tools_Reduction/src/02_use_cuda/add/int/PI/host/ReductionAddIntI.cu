#include "hip/hip_runtime.h"
#include "ReductionAddIntI.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KAddIntProtocoleI(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionAddIntI::ReductionAddIntI(const Grid& grid , int* ptrSum , bool isVerbose) :
	RunnableGPU(grid, "ReductionAddIntI-" + to_string(grid.threadCounts()), isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    // TODO ReductionAddIntI
    // MM pour ptrSumGM (oubliez pas initialisation)
    this->sizeSM = -1;

    // Tip:  Il y a une methode dedier pour malloquer un int cote device et l'initialiser a zero
    //
    //		GM::mallocInt0(&ptrSumGM);
    }

ReductionAddIntI::~ReductionAddIntI()
    {
    // TODO ReductionAddIntI
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionAddIntI::run()
    {
    // TODO ReductionAddIntI
    // appeler le kernel
    // recuperer le resulat coter host

    // Tip:  Il y a une methode dedier ramener coter host un int
    //
    //		GM::memcpyDtoH_int(ptrDestination, ptrSourceGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
