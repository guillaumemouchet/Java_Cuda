#include "hip/hip_runtime.h"
#include "ReductionAddIntI.h"

#include <assert.h>
#include <GM.h>
#include <GM_MemoryManagement.h>
#include <iostream>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KAddIntProtocoleI(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionAddIntI::ReductionAddIntI(const Grid& grid , int* ptrSum , bool isVerbose) :
	RunnableGPU(grid, "ReductionAddIntI-" + to_string(grid.threadCounts()), isVerbose), // classe parente
	ptrSum(ptrSum),
	dg(grid.dg),
	db(grid.db)
    {
    // TODO ReductionAddIntI
    // MM pour ptrSumGM (oubliez pas initialisation)
    this->sizeSM = sizeof(int) * grid.threadByBlock();
    // Tip:  Il y a une methode dedier pour malloquer un int cote device et l'initialiser a zero
    //
    GM::mallocInt0(&ptrSumGM);
    }

ReductionAddIntI::~ReductionAddIntI()
    {
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionAddIntI::run()
    {
    // TODO ReductionAddIntI
    // appeler le kernel
    // recuperer le resulat coter host
    KAddIntProtocoleI<<<dg, db, sizeSM>>>(ptrSumGM);
    // Tip:  Il y a une methode dedier ramener coter host un int
    //
    GM::memcpyDToH_int(ptrSum, ptrSumGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
