#include "hip/hip_runtime.h"
#include "ReductionAddIntII.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KAddIntProtocoleII(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionAddIntII::ReductionAddIntII(const Grid& grid , int* ptrSum,bool isVerbose) :
	RunnableGPU(grid, "ReductionAddIntII-" + to_string(grid.threadCounts()),isVerbose), // classe parente
	ptrSum(ptrSum),
	dg(grid.dg),
	db(grid.db)
    {
    // TODO ReductionAddIntII
    this->sizeSM = sizeof(int) * grid.threadByBlock();
    GM::mallocInt0(&ptrSumGM);

    }

ReductionAddIntII::~ReductionAddIntII()
    {
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionAddIntII::run()
    {
    KAddIntProtocoleII<<<dg, db, sizeSM>>>(ptrSumGM);
    GM::memcpyDToH_int(ptrSum, ptrSumGM);

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
