#include "hip/hip_runtime.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include "ReductionAdd.cu.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/


__global__ void KAddIntProtocoleII(int* ptrSumGM)
    {
    extern __shared__ int tabSM[];

    // ReductionIntraThread
    reductionIntraThread (tabSM);

    // ReductionAdd
    ReductionAdd::reduce(tabSM, ptrSumGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * TID partout en tabSM
 */
__device__ void reductionIntraThread(int* tabSM)
    {
    // TODO ReductionAddIntII
    const int TID_LOCAL = Thread2D::tidLocalBlock();
    const int TID = Thread2D::tid();
    tabSM[TID_LOCAL] = TID;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
