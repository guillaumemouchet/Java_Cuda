#include "hip/hip_runtime.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include "Reduction.cu.h"
#include "Lock.cu.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM);

// Operateur reduction
static __device__ int add(int x , int y);
static __device__ void addAtomicV1(int* ptrX , int y);
static __device__ void addAtomicV2(int* ptrX , int y);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void KIntProtocoleII(int* ptrSumGM)
    {
    // TODO ReductionIntII
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * TID partout en tabSM
 */
__device__ void reductionIntraThread(int* tabSM)
    {
    // TODO ReductionIntII
    }

/*----------------------------*\
|*	Operateur reduction    *|
 \*---------------------------*/

__device__ int add(int x , int y)
    {
    // TODO ReductionIntII
    }



/**
 * Utiliser la methode system : atomicAdd(pointeurDestination, valeurSource);
 */
__device__ void addAtomicV1(int* ptrX , int y)
    {
    // TODO ReductionIntII
    }

/**
 * Une alternative, moins performante, mais generalisable serait d'employer un lock
 * Tip : le Lock est implementer avec deux methodes atomic
 */
__device__ int volatile mutex = 0;	//variable global
__device__ void addAtomicV2(int* ptrX , int y)
    {
    Lock locker(&mutex);
    locker.lock();

    // TODO ReductionIntII

    locker.unlock();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

