#include "hip/hip_runtime.h"
#include "ReductionIntII.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KIntProtocoleII(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionIntII::ReductionIntII(const Grid& grid , int* ptrSum,bool isVerbose) :
	RunnableGPU(grid, "ReductionIntII-" + to_string(grid.threadCounts()),isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    // TODO ReductionIntII
    this->sizeSM = -1;
    }

ReductionIntII::~ReductionIntII()
    {
    // TODO ReductionIntII
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionIntII::run()
    {
    // TODO ReductionIntII
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
