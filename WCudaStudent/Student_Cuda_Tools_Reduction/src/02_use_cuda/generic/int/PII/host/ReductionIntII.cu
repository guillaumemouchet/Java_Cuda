#include "hip/hip_runtime.h"
#include "ReductionIntII.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KIntProtocoleII(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionIntII::ReductionIntII(const Grid& grid , int* ptrSum,bool isVerbose) :
	RunnableGPU(grid, "ReductionIntII-" + to_string(grid.threadCounts()),isVerbose), // classe parente
	ptrSum(ptrSum),
	dg(grid.dg),
	db(grid.db)
    {
    // TODO ReductionIntII
    this->sizeSM = sizeof(int) * grid.threadByBlock();
        GM::mallocInt0(&ptrSumGM);    }

ReductionIntII::~ReductionIntII()
    {
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionIntII::run()
    {
    KIntProtocoleII<<<dg, db, sizeSM>>>(ptrSumGM);
    GM::memcpyDToH_int(ptrSum, ptrSumGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
