#include "hip/hip_runtime.h"
#include "ReductionLongII.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KLongProtocoleII(long* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionLongII::ReductionLongII(const Grid& grid , long* ptrSum,bool isVerbose) :
	RunnableGPU(grid, "ReductionLongII-" + to_string(grid.threadCounts()),isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    // TODO ReductionLongII
    this->sizeSM = -1;
    }

ReductionLongII::~ReductionLongII()
    {
    // TODO ReductionLongII
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionLongII::run()
    {
    // TODO ReductionLongII
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
