#include "hip/hip_runtime.h"
#include "Thread1D_long.cu.h"
#include "cudas.h"

#include "Reduction.cu.h"
#include "Lock.cu.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(long* tabSM);

// Operateur reduction
static __device__ long add(long x , long y);
static __device__ void addAtomic(long* ptrX , long y);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * TID partout en tabSM
 */
__global__ void KLongProtocoleII(long* ptrSumGM)
    {
    // TODO ReductionLongII
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * TID partout en tabSM
 */
__device__ void reductionIntraThread(long* tabSM)
    {
    // Rappel : Dans le protocoleII on cherche a calculer
    //
    //			x=x+i avec i in [0,N]
    //
    //		ie la somme des entiers allant de 1 à N
    //
    //		Ces entiers sont mis dans tabSM, puis on somme le contenu des tabSM
    //
    // Technique : Chaque thread depose en tabSM, sont TID!
    //
    // Rappel	 : Chaque thread possede une et exactement une case en tabSM (relation 1-1)
    //
    // Warning   : L'utilisation du type long permet dans le protocoleII de sommer plus d'entier.
    //
    //		   Pour sommer plus d'entier, on prend plus de thread (cf relation 1-1) ci-dessus.
    //
    //             Plus de thread, c'est une grille plus grande, et dès lors le TID va devenir tres grand, plus grand que MAX_INT
    //
    //		Faux:		const int TID=Indice1D.tid(); // ne va pas fonctionner pour pour les grandes grids car max(TID)>MAX_INT
    //
    //		Juste (v1):	const long TID=((long)blockDim.x * (long) blockIdx.x)+ (long)threadIdx.x;
    //
    //		Juste (V2):	const long TID=Thread1D_long::tid();
    //
    //		Juste (V3):	const long TID=Thread2D_long::tid();
    //
    // Attention
    //
    //		Tout ceci est vrai seulement ici, dans le cadre du protocoleII

    // TODO ReductionLongII

    // pour TID       utiliser const long TID=Thread2D_long::tid();		// (nouvelle methode)
    // pour TID_LOCAL utiliser const int TID_LOCAL=Thread2D::tidLocal();	// (methode habituelle)
    }


/*----------------------------*\
|*	Operateur reduction    *|
 \*---------------------------*/

__device__ long add(long x , long y)
    {
    // TODO ReductionLongII
    }

/**
 * Utiliser la methode system, si elle existe
 *
 * 	atomicAdd(pointeurDestination, valeurSource)
 *
 * ou la technique du lock vu precedement!
 *
 * Question : atomicAdd pour les long existe?
 */
__device__ int volatile mutex = 0;	//variable global
__device__ void addAtomic(long* ptrX , long y)
    {
    Lock locker(&mutex);
    locker.lock();

    // TODO ReductionLongII

    locker.unlock();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

