#include "hip/hip_runtime.h"
#include <DomainMath3D_GPU.h>
#include <Indices.cu.h>
#include <Thread2D.cu.h>

#include "math/Damier3DMath_RGBA.h"

using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void damier3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , DomainMath3D domaineMath , float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * w nbPoint en x
 * h nbPoint en y
 */
__global__ void damier3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , DomainMath3D domaineMath , float t)
    {

    const int n = (int)t;
    Damier3DMath_RGBA damierMath(n);
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    const int WH = w * h;

    uchar4 color;
    float3 sommet;

    float x;
    float y;
    float DX;
    float DY;

    int vertexI;
    int vertexJ;

    domaineMath.delta(w, h, &DX, &DY);

    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &vertexI, &vertexJ); // update (vertexI, vertexJ)

	domaineMath.toXY(DX, DY, vertexI, vertexJ, &x, &y);

	damierMath.process(&sommet, &color, x, y, n); // update sommet

	tabVerticesColorGM[s] = color;
	tabVerticesXYZGM[s] = sommet;

	s += NB_THREAD;
	}

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
