#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "VideoSurface.h"

#include "GM.h"
#include "Hardware.h"
#include "Surface2D.h"

using std::cout;
using std::cerr;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void kSeuillage(hipSurfaceObject_t surf2d, uchar* tabPixelGM ,uint w , uint h, uchar seuil0255);
extern __global__ void KDilatation1( uchar* tabPixelGM,hipSurfaceObject_t surf2d , uint w , uint h );
extern __global__ void KDilatationN( uchar* tabPixelGM,hipSurfaceObject_t surf2d , uint w , uint h,int rayon );

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

VideoSurface::VideoSurface(const Grid& gridDilatation , IteratorImageGray_I* ptrIteratorImage , uchar seuil0255) :
	Animable_I<uchar>(gridDilatation, ptrIteratorImage->getW(), ptrIteratorImage->getH(), "Video_surface_uchar"), //
	//
	seuil0255(seuil0255),	//
	//
	ptrIteratorImage(ptrIteratorImage),	//
	//
	gridSeuillageBest(bestGridSeuillage()), //
	//
	wh(ptrIteratorImage->getWH()),	//
	//
	surface2d(ptrIteratorImage->getW(), ptrIteratorImage->getH(), CudaArrayType::SURFACE_LOAD_STORE)
    {
    // Tools
    this->t = 0;  // protected dans Animable

    this->sizeImage = sizeof(uchar) * wh;

    //video
	{
	this->tabPixelVideo = ptrIteratorImage->first();
	}
    }

VideoSurface::~VideoSurface()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void VideoSurface::process(uchar* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    // 1) copier tabPixelVideo dans tabPixelsGM
    // 2) seuillage
    // 3) dilatation

    // TODO
    }

/**
 * Override
 * Call periodicly by the API
 */
void VideoSurface::animationStep()
    {
    t++;

    this->tabPixelVideo = ptrIteratorImage->next();
    }

/*-------------------------*\
 |*	static		    *|
 \*-------------------------*/

Grid VideoSurface::bestGridSeuillage()
    {
    const int MP = Hardware::getMPCount();
    const int CORE_MP = Hardware::getCoreCountMP();

    // TODO
    assert(false);

    Grid gridConstrastor;

    return gridConstrastor;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

