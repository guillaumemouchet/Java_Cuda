#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"

#include "Calibreur.cu.h"
#include "Indices.cu.h"
#include "ifelse.cu.h"
#include "Edge.cu.h"

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * text2d 	: read
 * tabPixelGM   : write
 *
 * Note: fait par exemple disparaitre les points noirs dans les bandes blanches de la route (pour video autoroute)
 */
__global__ void KDilatation1(uchar* tabPixelGM , hipSurfaceObject_t surf2d , uint w , uint h)
    {
    // 1) Parcourir tous les pixels, y compris ceux du bord
    // 2) Voir si un pixel voisin est a 255, si oui on devient aussi a 255
    //
    // Truc   : grace a la surface pas besoin de gerer les bords de l'image de maniere specifique
    //
    // Rappel 1:  uchar pixels;
    //		 surf2Dread(&pixelS, surf2d, j, i);
    //
    // Rappel 2:
    //		uchar pixels= surf2Dread<uchar>(surf2d, j, i);
    //
    // Warning : (j,i) et non (i,j)

    // TODO KDilatation1

    }

/**
 * Amelioration:
 * 	on pourrait prendre un voisinage v8 avec une (ou plusieurs) aurole de plus, autour du pixels,
 *	pour faire disparaitre encore plus de zone noir dans la bande blanche, quitte a fire une errosion apres
 *	Voir KDilatationN ci-dessous
 */
__global__ void KDilatationN(uchar* tabPixelGM , hipSurfaceObject_t surf2d , uint w , uint h , int rayon)
    {
    // bonus
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

