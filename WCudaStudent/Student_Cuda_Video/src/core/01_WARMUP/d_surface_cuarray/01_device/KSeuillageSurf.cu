#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"

#include "Calibreur.cu.h"
#include "Indices.cu.h"
#include "ifelse.cu.h"

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/
/**
 * tabPixelGM   : read
 * surf2dGM     : write
 */
__global__ void kSeuillage(hipSurfaceObject_t surf2d , uchar* tabPixelGM , uint w , uint h , uchar seuil0255)
    {
    // 1) Chaque pixel devient 1 ou 255 selon si il est plus petit ou plus grand que le seuil0255
    // 2) Pour evitre la thread divergence avec un "if else standard", utiliser la methode "ifelse" de ifelse.cu.h
    //    Aller surtout regarder son implementation (f3 dans le include ci-dessus)
    //
    // Rappel : surf2Dwrite(pixelS, surf2d, j, i);  (j,i) et non (i,j)

    // TODO kSeuillage
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

