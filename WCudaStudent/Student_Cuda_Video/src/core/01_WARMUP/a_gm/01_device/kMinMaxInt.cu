#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"
#include "Reduction.cu.h"
#include "Lock.cu.h"

/*----------------------------------------------------------------------*\
 |*			Private 					*|
 \*---------------------------------------------------------------------*/

static __device__ void reductionIntraThread(uchar* tabPixelsGM,uint wh,int* tabMinSM,int* tabMaxSM );

/*----------------------------------------------------------------------*\
 |*			Tools 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	min/max		    *|
 \*-------------------------*/

__device__ void max(int* ptrA , int b)
    {
    *ptrA = MAX(*ptrA, b);
    }

__device__ void min(int* ptrA , int b)
    {
    *ptrA = MIN(*ptrA, b);
    }

/*-------------------------*\
 |*	min/max	ATOMIC	    *|
 \*-------------------------*/

__device__ void minAtomic(int* ptrA , int b)
    {
    atomicMin(ptrA, b);
    }

__device__ void maxAtomic(int* ptrA , int b)
    {
    atomicMax(ptrA, b);
    }

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * char to int car atomicMin n'existe pas pour uchar
 */
__global__ void kMinMaxInt(uchar* tabPixelsGM , uint wh , int* ptrMinGM , int* ptrMaxGM)
    {
    // 2 tableaux en SM depuis un 1 tableau 2x plus grand
    extern __shared__ int tabSM[];
    int* tabMinSM = tabSM;
    int* tabMaxSM = tabSM + Thread2D::nbThreadBlock(); // car il y a autant de case en SM que de thread par block

	    // 1) reduction IntraThread
	    // 2) reduction (inra-block et inter-block) de tabMinSM : resultat dans ptrMinGM
	    // 3) reduction (inra-block et inter-block) de tabMaxSM : resultat dans ptrMaxGM

	    // TODO kMinMaxInt
    }

/**
 * char to int car atomicMin n'existe pas pour uchar
 */
__device__ void reductionIntraThread(uchar* tabPixelsGM , uint wh , int* tabMinSM , int* tabMaxSM)
    {
    // on travaille en int, car atomicMin/atomicMax n'existe pas en uchar (cuda11)
    int min = 255;
    int max = 0;

    // TODO
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

