#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"

#include "Calibreur.cu.h"

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void kConstrastor(uchar* tabPixelsGM , uint wh , int* ptrMinGM , int* ptrMaxGM)
    {
    // Goal  : [min,max]-> [0,255]
    //
    // Tools : Calibreur<float> calibreur(Interval<float> depart, Interval<float> arriver)
    //	       calibreur.calibrer(&toCalibrer);
    //	       travailler en float pour la calibration
    //
    // Astuce:  Interval<float> intervalArriver(0, 255.9f); // 255.9f sinon on obtient jamais 255, mais 254 comme max

    float min = *ptrMinGM;
    float max = *ptrMaxGM;

    // TODO kConstrastor
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

