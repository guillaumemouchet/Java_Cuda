#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "VideoTextureGM.h"
#include "GM.h"
#include "Stream.h"
#include "Hardware.h"
#include "Limits.h"
#include "Texture2D_GM.h"

using std::cout;
using std::cerr;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void kMinMaxInt(uchar* tabPixelsGM , uint wh , int* ptrDevMin, int* ptrDevMax);
extern __global__ void kConstrastorTex(uchar* tabPixelsGM,hipTextureObject_t tex2dGM , uint w , uint h , int* ptrMinGM , int* ptrMaxGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

VideoTextureGM::VideoTextureGM(const Grid& gridMinMax , IteratorImageGray_I* ptrIteratorImage) :
	Animable_I<uchar>(gridMinMax, ptrIteratorImage->getW(), ptrIteratorImage->getH(), "Video_texture_gm_uchar"), //
	//
	ptrIteratorImage(ptrIteratorImage), //
	//
	gridMinMax(gridMinMax), //
	gridConstrastor(bestGridConstrastor()), //
	//
	wh(ptrIteratorImage->getWH()), //
	//
	texture2D_GM(ptrIteratorImage->getW(), ptrIteratorImage->getH()) // NEW
    {
    // Tools
    this->t = 0; // protected dans Animable

    this->sizeImage = sizeof(uchar) * wh;

    //video
	{
	this->tabPixelVideo = ptrIteratorImage->first();
	}

    // Tools : minmax
	{
	// host
	this->tabMinMax = new int[2];

	// device
	this->sizeMinMax = 2 * sizeof(int); // 2 car min et max
	this->sizeSM = gridMinMax.threadByBlock() * sizeMinMax;

	GM::malloc(&tabMinMaxGM, sizeMinMax);

	this->ptrMinGM = tabMinMaxGM;
	this->ptrMaxGM = tabMinMaxGM + 1;
	}
    }

VideoTextureGM::~VideoTextureGM()
    {
    // MM
	{
	GM::free(tabMinMaxGM);
	}

    delete[] tabMinMax;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void VideoTextureGM::process(uchar* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    // 1) Copier tabPixelVideo dans tabPixelsGM
    // 2) Associer tabPixelsGM a la texture :  texture2D_GM.see(tabPixelsGM);  // NEW
    // 3) Calculer le min et max de tabPixelsGM
    // 4) Contraster tabPixelsGM
    // 5) Verifier que le contrastage a fonctionner, ie que (min,max)=(0,255)

    // TODO

    minMax(tabPixelsGM, "Before");
    constrator(tabPixelsGM);
    }

/**
 * Override
 * Call periodicly by the API
 */
void VideoTextureGM::animationStep()
    {
    t++;

    this->tabPixelVideo = ptrIteratorImage->next();
    }

/*-------------------------*\
 |*	private		    *|
 \*-------------------------*/

void VideoTextureGM::constrator(uchar* tabPixelsGM)
    {
    int min = tabMinMax[0];
    int max = tabMinMax[1];

    if (min != max || (min != 0 && max != 255))
	{
	// TODO
	assert(false);

	// Check (benchmark && bruteforce && rendu final : laisser activer)
	    {
	    minMax(tabPixelsGM, "After ");

	    assert(tabMinMax[0] == 0);
	    assert(tabMinMax[1] == 255);
	    }
	}
    }

void VideoTextureGM::minMax(uchar* tabPixelsGM , string title)
    {
    // Warning : reinitialiser a chaque fois!
	{
	tabMinMax[0] = Limits::MAX_INT; 	// init min
	tabMinMax[1] = 0;			// init max

	GM::memcpyHToD(tabMinMaxGM, tabMinMax, sizeMinMax); // on copie le min et max d'un coup!
	}

    // 1) calculer min/max sur le device dans tabMinMaxGM
    // 2) ramener  min/max sur le host   dans tabMinMax

    // TODO
    assert(false);

    // debug
    //	{
    //	cout << title << " : (min,max)=(" << tabMinMax[0] << "," << tabMinMax[1] << ")" << endl;
    //	}
    }

/*-------------------------*\
 |*	static		    *|
 \*-------------------------*/

Grid VideoTextureGM::bestGridConstrastor()
    {
    const int MP = Hardware::getMPCount();
    const int CORE_MP = Hardware::getCoreCountMP();

    // TODO
    assert(false);
    Grid gridConstrastor;

    return gridConstrastor;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

