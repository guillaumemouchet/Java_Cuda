#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"

#include "Calibreur.cu.h"
#include "Indices.cu.h"

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * tex2dGM 	: read
 * tabPixelsGM  : write
 */
__global__ void kConstrastorTex(uchar* tabPixelsGM , hipTextureObject_t tex2dGM , uint w , uint h , int* ptrMinGM , int* ptrMaxGM)
    {
    // Goal  : [min,max]-> [0,255]
    //
    // Tools : Calibreur<float> calibreur(Interval<float> depart, Interval<float> arriver)
    //	       calibreur.calibrer(&toCalibrer);
    //	       travailler en float pour la calibration
    //
    // Astuce:  Interval<float> intervalArriver(0, 255.9f); // 255.9f sinon on obtient jamais 255, mais 254 comme max

    float min = *ptrMinGM;
    float max = *ptrMaxGM;

    // TODO kConstrastorTex
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

