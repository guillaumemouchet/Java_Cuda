#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Maths.h"

#include "Indices.cu.h"

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * <pre>
 * source : 	tex2dVideo
 * destination :tabPixelsGM
 * </pre>
 */
__global__ void kContourTex(hipTextureObject_t tex2dVideo , uchar* tabPixelsGM , uint w , uint h)
    {
    // on lit dans tex2dVideo
    // Youpie : 	Pas besoin de gerer les bords car on a une texture
    // Rappel :		uchar pixels = tex2D < uchar > (tex2dVideo, j,i);
    // warning: 	tex2D(j,i) et non tex2D(i,j)

    // TODO kContourTex
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

