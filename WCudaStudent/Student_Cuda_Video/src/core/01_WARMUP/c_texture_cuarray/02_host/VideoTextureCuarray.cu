#include "hip/hip_runtime.h"
#include <iostream>

#include "VideoTextureCuarray.h"
#include "Texture2D.h"

using std::cout;
using std::cerr;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/**
 * <pre>
 * source : 	tex2dVideo
 * destination :tabPixelsGM
 * </pre>
 */
extern __global__ void kContourTex(hipTextureObject_t tex2dVideo,uchar* tabPixelsGM , uint w, uint h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

VideoTextureCuarray::VideoTextureCuarray(const Grid& grid , IteratorImageGray_I* ptrIteratorImage) :
	Animable_I<uchar>(grid, ptrIteratorImage->getW(), ptrIteratorImage->getH(), "Video_contour_Texture_cuarray_uchar"), //
	//
	ptrIteratorImage(ptrIteratorImage),	//
	//
	texture2DVideo(ptrIteratorImage->getW(), ptrIteratorImage->getH())
    {
    // animation
	{
	this->t = 0;  // protected dans Animable
	}

    //video
	{
	this->tabPixelVideo = ptrIteratorImage->first();
	}
    }

VideoTextureCuarray::~VideoTextureCuarray()
    {
    //rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void VideoTextureCuarray::process(uchar* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    // 1) copier tabPixelVideo dans texture2DVideo
    // 2) appeler le kernel de contour

    // TODO
    }

/**
 * Override
 * Call periodicly by the API
 */
void VideoTextureCuarray::animationStep()
    {
    t++;

    this->tabPixelVideo = ptrIteratorImage->next();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

